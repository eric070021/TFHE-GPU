#include "hip/hip_runtime.h"
#include "bootstrapping.cuh"

#ifndef CUDA_CHECK_AND_EXIT
#    define CUDA_CHECK_AND_EXIT(error)                                                                      \
        {                                                                                                   \
            auto status = static_cast<hipError_t>(error);                                                  \
            if (status != hipSuccess) {                                                                    \
                std::cout << hipGetErrorString(status) << " " << __FILE__ << ":" << __LINE__ << std::endl; \
                std::exit(status);                                                                          \
            }                                                                                               \
        }
#endif // CUDA_CHECK_AND_EXIT

namespace lbcrypto {

__device__ inline void ModSubFastEq_CUDA(uint64_t &a, const uint64_t &b, const uint64_t &modulus) {
        if (a >= b) {
            a -= b;
        }
        else {
            a += (modulus - b);
        }
}

__device__ inline uint64_t RoundqQ_CUDA(const uint64_t &v, const uint64_t &q, const uint64_t &Q) {
    return static_cast<uint64_t>(floor(0.5 + static_cast<double>(v) * static_cast<double>(q) / static_cast<double>(Q))) % q;
}

__global__ void MKMSwitchKernel(uint64_t* ctExt_CUDA, uint64_t* keySwitchingkey_CUDA, uint64_t *paramsMKM_CUDA){
    /* GPU Parameters Set */
    uint32_t tid = ThisThreadRankInBlock();
    uint32_t bdim = ThisBlockSize();

    /* HE Parameters Set */
    uint64_t n              = paramsMKM_CUDA[0];
    uint64_t N              = paramsMKM_CUDA[1];
    uint64_t Q              = paramsMKM_CUDA[3];
    uint64_t baseKS         = paramsMKM_CUDA[4];
    uint64_t digitCountKS   = paramsMKM_CUDA[5];
    uint64_t Q1             = paramsMKM_CUDA[6];
    uint64_t Q2             = paramsMKM_CUDA[7];

    /* First Modswitch */
    for (size_t i = tid; i <= N; i += bdim)
        ctExt_CUDA[i] = RoundqQ_CUDA(ctExt_CUDA[i], Q1, Q);
    __syncthreads();

    /* KeySwitch */
    extern __shared__ uint64_t ctKS[];
    for(uint32_t i = tid; i <= n; i += bdim){
        ctKS[i] = 0;
    }
    __syncthreads();
    // a
    for (uint32_t i = 0; i < N; ++i) {
        uint64_t atmp = ctExt_CUDA[i];
        for (uint32_t j = 0; j < digitCountKS; ++j, atmp /= baseKS) {
            uint64_t a0 = (atmp % baseKS);
            for (uint32_t k = tid; k < n; k += bdim)
                ModSubFastEq_CUDA(ctKS[k], keySwitchingkey_CUDA[i*baseKS*digitCountKS*n + a0*digitCountKS*n + j*n + k], Q1);
        }
    }
    __syncthreads();
    // b
    if(tid == 0){
        ctKS[n] = ctExt_CUDA[N];
        for (uint32_t i = 0; i < N; ++i) {
            uint64_t atmp = ctExt_CUDA[i];
            for (uint32_t j = 0; j < digitCountKS; ++j, atmp /= baseKS) {
                uint64_t a0 = (atmp % baseKS);
                ModSubFastEq_CUDA(ctKS[n], keySwitchingkey_CUDA[N*baseKS*digitCountKS*n + i*baseKS*digitCountKS + a0*digitCountKS + j], Q1);
            }
        }
    }
    __syncthreads();

    /* Second Modswitch */
    for (size_t i = tid; i <= n; i += bdim)
        ctKS[i] = RoundqQ_CUDA(ctKS[i], Q2, Q1);
    __syncthreads();

    /* Copy ctKS to ctExt_CUDA */
    for(uint32_t i = tid; i <= n; i += bdim){
        ctExt_CUDA[i] = ctKS[i];
    }
    __syncthreads();
}

template<class FFT, class IFFT>
__launch_bounds__(FFT::max_threads_per_block)
__global__ void bootstrappingMultiBlock(Complex_d* acc_CUDA, Complex_d* ct_CUDA, Complex_d* dct_CUDA, uint64_t* a_CUDA, 
        Complex_d* monomial_CUDA, Complex_d* twiddleTable_CUDA, uint64_t* params_CUDA, Complex_d* GINX_bootstrappingKey_CUDA){
    
    /* GPU Parameters Set */
    cg::grid_group grid = cg::this_grid();
    uint32_t tid = ThisThreadRankInBlock(); // thread id in block
    uint32_t bid = grid.block_rank(); // block id in grid
    uint32_t gtid = grid.thread_rank(); // global thread id
    uint32_t bdim = ThisBlockSize(); // size of block
    uint32_t gdim = grid.num_threads(); // number of threads in grid

    /* HE Parameters Set */
    uint64_t M            = params_CUDA[0] << 1;
    uint64_t N            = params_CUDA[0];
    uint64_t NHalf        = N >> 1;
    uint64_t n            = params_CUDA[1];
    uint64_t Q            = params_CUDA[2];
    uint64_t QHalf        = params_CUDA[2] >> 1;
    uint64_t digitsG2     = params_CUDA[3];
    uint64_t baseG        = params_CUDA[4];
    int32_t gBits = static_cast<int32_t>(log2(static_cast<double>(baseG)));
    int32_t gBitsMaxBits = 64 - gBits;
    uint32_t RGSW_size = digitsG2 * 2 * NHalf;

    /* cufftdx variables */
    using complex_type = typename FFT::value_type;
    const unsigned int local_fft_id = threadIdx.y;
    const unsigned int offset = cufftdx::size_of<FFT>::value * (blockIdx.x * FFT::ffts_per_block + local_fft_id);
    extern __shared__ complex_type shared_mem[];
    complex_type thread_data[FFT::storage_size];     
    
    /* 2 times Forward FFT */
    if(bid == 0){
        // Load data from shared memory to registers
        {
            unsigned int index = offset + threadIdx.x;
            unsigned int twist_idx = threadIdx.x;
            for (unsigned i = 0; i < FFT::elements_per_thread; i++) {
                // twisting
                acc_CUDA[index] = hipCmul(acc_CUDA[index], twiddleTable_CUDA[twist_idx]);
                thread_data[i] = complex_type {acc_CUDA[index].x, acc_CUDA[index].y};
                // FFT::stride shows how elements from a single FFT should be split between threads
                index += FFT::stride;
                twist_idx += FFT::stride;
            }
        }

        FFT().execute(thread_data, shared_mem);

        // Save results
        {
            unsigned int index = offset + threadIdx.x;
            for (unsigned i = 0; i < FFT::elements_per_thread; i++) {
                acc_CUDA[index] = make_hipDoubleComplex(thread_data[i].x, thread_data[i].y);
                // FFT::stride shows how elements from a single FFT should be split between threads
                index += FFT::stride;
            }
        }
    }
    grid.sync();
    
    for(uint32_t round = 0; round < n; ++round){
        /* Copy acc_CUDA to ct_CUDA */
        for(uint32_t i = gtid; i < N; i += gdim){
            ct_CUDA[i] = acc_CUDA[i];
        }
        grid.sync();

        /* 2 times Inverse IFFT */
        if(bid == 0){
            // Load data from shared memory to registers
            {
                unsigned int index = offset + threadIdx.x;
                for (unsigned i = 0; i < IFFT::elements_per_thread; i++) {
                    thread_data[i] = complex_type {ct_CUDA[index].x, ct_CUDA[index].y};
                    // FFT::stride shows how elements from a single FFT should be split between threads
                    index += IFFT::stride;
                }
            }

            // Scale values
            double scale = 1.0 / cufftdx::size_of<IFFT>::value;
            for (unsigned int i = 0; i < IFFT::elements_per_thread; i++) {
                thread_data[i].x *= scale;
                thread_data[i].y *= scale;
            }

            IFFT().execute(thread_data, shared_mem);
        
            // Save results
            {
                unsigned int index = offset + threadIdx.x;
                unsigned int twist_idx = threadIdx.x;
                for (unsigned i = 0; i < IFFT::elements_per_thread; i++) {
                    ct_CUDA[index].x = thread_data[i].x;
                    ct_CUDA[index].y = thread_data[i].y;
                    // twisting
                    ct_CUDA[index] = hipCmul(ct_CUDA[index], twiddleTable_CUDA[twist_idx + NHalf]);
                    // Round to INT128 and MOD
                    ct_CUDA[index].x = static_cast<double>(static_cast<__int128_t>(rint(ct_CUDA[index].x)) % static_cast<__int128_t>(Q));
                    if (ct_CUDA[index].x < 0)
                        ct_CUDA[index].x += static_cast<double>(Q);
                    if (ct_CUDA[index].x >= QHalf)
                        ct_CUDA[index].x -= static_cast<double>(Q);
                    ct_CUDA[index].y = static_cast<double>(static_cast<__int128_t>(rint(ct_CUDA[index].y)) % static_cast<__int128_t>(Q));
                    if (ct_CUDA[index].y < 0)
                        ct_CUDA[index].y += static_cast<double>(Q);
                    if (ct_CUDA[index].y >= QHalf)
                        ct_CUDA[index].y -= static_cast<double>(Q);
                    // IFFT::stride shows how elements from a single FFT should be split between threads
                    index += IFFT::stride;
                    twist_idx += IFFT::stride;
                }
            }
        }
        grid.sync();

        /* SignedDigitDecompose */
        // polynomial from a
        for (size_t k = gtid; k < NHalf; k += gdim) {
            int64_t d0 = static_cast<int64_t>(ct_CUDA[k].x);
            int64_t d1 = static_cast<int64_t>(ct_CUDA[k].y);

            for (size_t l = 0; l < digitsG2; l += 2) {
                int64_t r0 = (d0 << gBitsMaxBits) >> gBitsMaxBits;
                d0 = (d0 - r0) >> gBits;
                if (r0 < 0)
                    r0 += static_cast<int64_t>(Q);
                if (r0 >= QHalf)
                    r0 -= static_cast<int64_t>(Q);
                dct_CUDA[l*NHalf + k].x = static_cast<double>(r0);

                int64_t r1 = (d1 << gBitsMaxBits) >> gBitsMaxBits;
                d1 = (d1 - r1) >> gBits;
                if (r1 < 0)
                    r1 += static_cast<int64_t>(Q);
                if (r1 >= QHalf)
                    r1 -= static_cast<int64_t>(Q);
                dct_CUDA[l*NHalf + k].y = static_cast<double>(r1);
            }
        }

        // polynomial from b
        for (size_t k = gtid + NHalf; k < N; k += gdim) {
            int64_t d0 = static_cast<int64_t>(ct_CUDA[k].x);
            int64_t d1 = static_cast<int64_t>(ct_CUDA[k].y);

            for (size_t l = 0; l < digitsG2; l += 2) {
                int64_t r0 = (d0 << gBitsMaxBits) >> gBitsMaxBits;
                d0 = (d0 - r0) >> gBits;
                if (r0 < 0)
                    r0 += static_cast<int64_t>(Q);
                if (r0 >= QHalf)
                    r0 -= static_cast<int64_t>(Q);
                dct_CUDA[l*NHalf + k].x = static_cast<double>(r0);

                int64_t r1 = (d1 << gBitsMaxBits) >> gBitsMaxBits;
                d1 = (d1 - r1) >> gBits;
                if (r1 < 0)
                    r1 += static_cast<int64_t>(Q);
                if (r1 >= QHalf)
                    r1 -= static_cast<int64_t>(Q);
                dct_CUDA[l*NHalf + k].y = static_cast<double>(r1);
            }
        }
        grid.sync();

        /* digitsG2 times Forward FFT */
        // Load data from shared memory to registers
        {
            unsigned int index = offset + threadIdx.x;
            unsigned int twist_idx = threadIdx.x;
            for (unsigned i = 0; i < FFT::elements_per_thread; i++) {
                // twisting
                dct_CUDA[index] = hipCmul(dct_CUDA[index], twiddleTable_CUDA[twist_idx]);
                thread_data[i] = complex_type {dct_CUDA[index].x, dct_CUDA[index].y};
                // FFT::stride shows how elements from a single FFT should be split between threads
                index += FFT::stride;
                twist_idx += FFT::stride;
            }
        }

        FFT().execute(thread_data, shared_mem);

        // Save results
        {
            unsigned int index = offset + threadIdx.x;
            for (unsigned i = 0; i < FFT::elements_per_thread; i++) {
                dct_CUDA[index] = make_hipDoubleComplex(thread_data[i].x, thread_data[i].y);
                // FFT::stride shows how elements from a single FFT should be split between threads
                index += FFT::stride;
            }
        }
        grid.sync();

        /* Obtain monomial */
        // First obtain both monomial(index) for sk = 1 and monomial(-index) for sk = -1
        auto aNeg         = (M - a_CUDA[round]) % M;
        uint64_t indexPos = a_CUDA[round];
        uint64_t indexNeg = aNeg;
        // index is in range [0,m] - so we need to adjust the edge case when
        // index = m to index = 0
        if (indexPos == M)
            indexPos = 0;
        if (indexNeg == M)
            indexNeg = 0;
        
        /* ACC times Bootstrapping key and monomial */
        /* multiply with ek0 */
        // polynomial a
        for (uint32_t i = gtid; i < NHalf; i += gdim){
            ct_CUDA[i] = make_hipDoubleComplex(0, 0);
            for (uint32_t l = 0; l < digitsG2; ++l){
                ct_CUDA[i].x = fma(dct_CUDA[l*NHalf + i].x, GINX_bootstrappingKey_CUDA[round*RGSW_size + (l << 1)*NHalf + i].x, ct_CUDA[i].x);
                ct_CUDA[i].x = fma(-dct_CUDA[l*NHalf + i].y, GINX_bootstrappingKey_CUDA[round*RGSW_size + (l << 1)*NHalf + i].y, ct_CUDA[i].x);
                ct_CUDA[i].y = fma(dct_CUDA[l*NHalf + i].x, GINX_bootstrappingKey_CUDA[round*RGSW_size + (l << 1)*NHalf + i].y, ct_CUDA[i].y);
                ct_CUDA[i].y = fma(dct_CUDA[l*NHalf + i].y, GINX_bootstrappingKey_CUDA[round*RGSW_size + (l << 1)*NHalf + i].x, ct_CUDA[i].y);
            }
        }
        // polynomial b
        for (uint32_t i = gtid; i < NHalf; i += gdim){
            ct_CUDA[NHalf + i] = make_hipDoubleComplex(0.0, 0.0);
            for (uint32_t l = 0; l < digitsG2; ++l){
                ct_CUDA[NHalf + i].x = fma(dct_CUDA[l*NHalf + i].x, GINX_bootstrappingKey_CUDA[round*RGSW_size + ((l << 1) + 1)*NHalf + i].x, ct_CUDA[NHalf + i].x);
                ct_CUDA[NHalf + i].x = fma(-dct_CUDA[l*NHalf + i].y, GINX_bootstrappingKey_CUDA[round*RGSW_size + ((l << 1) + 1)*NHalf + i].y, ct_CUDA[NHalf + i].x);
                ct_CUDA[NHalf + i].y = fma(dct_CUDA[l*NHalf + i].x, GINX_bootstrappingKey_CUDA[round*RGSW_size + ((l << 1) + 1)*NHalf + i].y, ct_CUDA[NHalf + i].y);
                ct_CUDA[NHalf + i].y = fma(dct_CUDA[l*NHalf + i].y, GINX_bootstrappingKey_CUDA[round*RGSW_size + ((l << 1) + 1)*NHalf + i].x, ct_CUDA[NHalf + i].y);
            }
        }
        grid.sync();
        /* multiply with postive monomial */
        // polynomial a
        for (uint32_t i = gtid; i < NHalf; i += gdim){
            acc_CUDA[i].x = fma(ct_CUDA[i].x, monomial_CUDA[indexPos*NHalf + i].x, acc_CUDA[i].x);
            acc_CUDA[i].x = fma(-ct_CUDA[i].y, monomial_CUDA[indexPos*NHalf + i].y, acc_CUDA[i].x);
            acc_CUDA[i].y = fma(ct_CUDA[i].x, monomial_CUDA[indexPos*NHalf + i].y, acc_CUDA[i].y);
            acc_CUDA[i].y = fma(ct_CUDA[i].y, monomial_CUDA[indexPos*NHalf + i].x, acc_CUDA[i].y);
        }
        // polynomial b
        for (uint32_t i = gtid; i < NHalf; i += gdim){
            acc_CUDA[NHalf + i].x = fma(ct_CUDA[NHalf + i].x, monomial_CUDA[indexPos*NHalf + i].x, acc_CUDA[NHalf + i].x);
            acc_CUDA[NHalf + i].x = fma(-ct_CUDA[NHalf + i].y, monomial_CUDA[indexPos*NHalf + i].y, acc_CUDA[NHalf + i].x);
            acc_CUDA[NHalf + i].y = fma(ct_CUDA[NHalf + i].x, monomial_CUDA[indexPos*NHalf + i].y, acc_CUDA[NHalf + i].y);
            acc_CUDA[NHalf + i].y = fma(ct_CUDA[NHalf + i].y, monomial_CUDA[indexPos*NHalf + i].x, acc_CUDA[NHalf + i].y);
        }        
        grid.sync();

        /* multiply with ek1 */
        // polynomial a
        for (uint32_t i = gtid; i < NHalf; i += gdim){
            ct_CUDA[i] = make_hipDoubleComplex(0, 0);
            for (uint32_t l = 0; l < digitsG2; ++l){
                ct_CUDA[i].x = fma(dct_CUDA[l*NHalf + i].x, GINX_bootstrappingKey_CUDA[n*RGSW_size + round*RGSW_size + (l << 1)*NHalf + i].x, ct_CUDA[i].x);
                ct_CUDA[i].x = fma(-dct_CUDA[l*NHalf + i].y, GINX_bootstrappingKey_CUDA[n*RGSW_size + round*RGSW_size + (l << 1)*NHalf + i].y, ct_CUDA[i].x);
                ct_CUDA[i].y = fma(dct_CUDA[l*NHalf + i].x, GINX_bootstrappingKey_CUDA[n*RGSW_size + round*RGSW_size + (l << 1)*NHalf + i].y, ct_CUDA[i].y);
                ct_CUDA[i].y = fma(dct_CUDA[l*NHalf + i].y, GINX_bootstrappingKey_CUDA[n*RGSW_size + round*RGSW_size + (l << 1)*NHalf + i].x, ct_CUDA[i].y);
            }
        }
        // polynomial b
        for (uint32_t i = gtid; i < NHalf; i += gdim){
            ct_CUDA[NHalf + i] = make_hipDoubleComplex(0.0, 0.0);
            for (uint32_t l = 0; l < digitsG2; ++l){
                ct_CUDA[NHalf + i].x = fma(dct_CUDA[l*NHalf + i].x, GINX_bootstrappingKey_CUDA[n*RGSW_size + round*RGSW_size + ((l << 1) + 1)*NHalf + i].x, ct_CUDA[NHalf + i].x);
                ct_CUDA[NHalf + i].x = fma(-dct_CUDA[l*NHalf + i].y, GINX_bootstrappingKey_CUDA[n*RGSW_size + round*RGSW_size + ((l << 1) + 1)*NHalf + i].y, ct_CUDA[NHalf + i].x);
                ct_CUDA[NHalf + i].y = fma(dct_CUDA[l*NHalf + i].x, GINX_bootstrappingKey_CUDA[n*RGSW_size + round*RGSW_size + ((l << 1) + 1)*NHalf + i].y, ct_CUDA[NHalf + i].y);
                ct_CUDA[NHalf + i].y = fma(dct_CUDA[l*NHalf + i].y, GINX_bootstrappingKey_CUDA[n*RGSW_size + round*RGSW_size + ((l << 1) + 1)*NHalf + i].x, ct_CUDA[NHalf + i].y);
            }
        }
        grid.sync();
        /* multiply with negative monomial */
        // polynomial a
        for (uint32_t i = gtid; i < NHalf; i += gdim){
            acc_CUDA[i].x = fma(ct_CUDA[i].x, monomial_CUDA[indexNeg*NHalf + i].x, acc_CUDA[i].x);
            acc_CUDA[i].x = fma(-ct_CUDA[i].y, monomial_CUDA[indexNeg*NHalf + i].y, acc_CUDA[i].x);
            acc_CUDA[i].y = fma(ct_CUDA[i].x, monomial_CUDA[indexNeg*NHalf + i].y, acc_CUDA[i].y);
            acc_CUDA[i].y = fma(ct_CUDA[i].y, monomial_CUDA[indexNeg*NHalf + i].x, acc_CUDA[i].y);
        }
        // polynomial b
        for (uint32_t i = gtid; i < NHalf; i += gdim){
            acc_CUDA[NHalf + i].x = fma(ct_CUDA[NHalf + i].x, monomial_CUDA[indexNeg*NHalf + i].x, acc_CUDA[NHalf + i].x);
            acc_CUDA[NHalf + i].x = fma(-ct_CUDA[NHalf + i].y, monomial_CUDA[indexNeg*NHalf + i].y, acc_CUDA[NHalf + i].x);
            acc_CUDA[NHalf + i].y = fma(ct_CUDA[NHalf + i].x, monomial_CUDA[indexNeg*NHalf + i].y, acc_CUDA[NHalf + i].y);
            acc_CUDA[NHalf + i].y = fma(ct_CUDA[NHalf + i].y, monomial_CUDA[indexNeg*NHalf + i].x, acc_CUDA[NHalf + i].y);
        }        
        grid.sync();
    }

    /* 2 times Inverse IFFT */
    if(bid == 0){
        // Load data from shared memory to registers
        {
            unsigned int index = offset + threadIdx.x;
            for (unsigned i = 0; i < IFFT::elements_per_thread; i++) {
                thread_data[i] = complex_type {acc_CUDA[index].x, acc_CUDA[index].y};
                // FFT::stride shows how elements from a single FFT should be split between threads
                index += IFFT::stride;
            }
        }

        // Scale values
        double scale = 1.0 / cufftdx::size_of<IFFT>::value;
        for (unsigned int i = 0; i < IFFT::elements_per_thread; i++) {
            thread_data[i].x *= scale;
            thread_data[i].y *= scale;
        }

        IFFT().execute(thread_data, shared_mem);
    
        // Save results
        {
            unsigned int index = offset + threadIdx.x;
            unsigned int twist_idx = threadIdx.x;
            for (unsigned i = 0; i < IFFT::elements_per_thread; i++) {
                acc_CUDA[index].x = thread_data[i].x;
                acc_CUDA[index].y = thread_data[i].y;
                // twisting
                acc_CUDA[index] = hipCmul(acc_CUDA[index], twiddleTable_CUDA[twist_idx + NHalf]);
                // Round to INT128 and MOD
                acc_CUDA[index].x = static_cast<double>(static_cast<__int128_t>(rint(acc_CUDA[index].x)) % static_cast<__int128_t>(Q));
                if (acc_CUDA[index].x < 0)
                    acc_CUDA[index].x += static_cast<double>(Q);
                acc_CUDA[index].y = static_cast<double>(static_cast<__int128_t>(rint(acc_CUDA[index].y)) % static_cast<__int128_t>(Q));
                if (acc_CUDA[index].y < 0)
                    acc_CUDA[index].y += static_cast<double>(Q);
                // IFFT::stride shows how elements from a single FFT should be split between threads
                index += IFFT::stride;
                twist_idx += FFT::stride;
            }
        }
    }
    grid.sync();

    /****************************************
    * the accumulator result is encrypted w.r.t. the transposed secret key
    * we can transpose "a" to get an encryption under the original secret key z
    * z = (z0, −zq/2−1, . . . , −z1)
    *****************************************/
    /* Copy acc_CUDA to ct_CUDA */
    for(uint32_t i = gtid; i < NHalf; i += gdim){
        ct_CUDA[i] = acc_CUDA[i];
    }
    grid.sync();

    for(uint32_t i = gtid+1; i < NHalf; i += gdim){
        acc_CUDA[i].x = static_cast<double>((Q - static_cast<uint64_t>(ct_CUDA[NHalf - i].y)));
        acc_CUDA[i].y = static_cast<double>((Q - static_cast<uint64_t>(ct_CUDA[NHalf - i].x)));
    }
    if(gtid == 0) acc_CUDA[0].y = static_cast<double>((Q - static_cast<uint64_t>(ct_CUDA[0].y)));
    grid.sync();
}

template<class FFT, class IFFT>
__launch_bounds__(FFT::max_threads_per_block)
__global__ void bootstrappingSingleBlock(Complex_d* acc_CUDA, Complex_d* ct_CUDA, Complex_d* dct_CUDA, uint64_t* a_CUDA, 
        Complex_d* monomial_CUDA, Complex_d* twiddleTable_CUDA, uint64_t* params_CUDA, Complex_d* GINX_bootstrappingKey_CUDA){
    
    /* GPU Parameters Set */
    uint32_t tid = ThisThreadRankInBlock();
    uint32_t bdim = ThisBlockSize();

    /* HE Parameters Set */
    uint64_t M            = params_CUDA[0] << 1;
    uint64_t N            = params_CUDA[0];
    uint64_t NHalf        = N >> 1;
    uint64_t n            = params_CUDA[1];
    uint64_t Q            = params_CUDA[2];
    uint64_t QHalf        = params_CUDA[2] >> 1;
    uint64_t digitsG2     = params_CUDA[3];
    uint64_t baseG        = params_CUDA[4];
    int32_t gBits = static_cast<int32_t>(log2(static_cast<double>(baseG)));
    int32_t gBitsMaxBits = 64 - gBits;
    uint32_t RGSW_size = digitsG2 * 2 * NHalf;
    uint32_t syncNum      = static_cast<uint32_t>(params_CUDA[5]); // number of synchronization (cufftdx)

    /* cufftdx variables */
    using complex_type = typename FFT::value_type;
    const unsigned int local_fft_id = threadIdx.y;
    const unsigned int offset = cufftdx::size_of<FFT>::value * (blockIdx.x * FFT::ffts_per_block + local_fft_id);
    extern __shared__ complex_type shared_mem[];
    complex_type thread_data[FFT::storage_size];     
    
    /* 2 times Forward FFT */
    if(threadIdx.y < 2){
        // Load data from shared memory to registers
        {
            unsigned int index = offset + threadIdx.x;
            unsigned int twist_idx = threadIdx.x;
            for (unsigned i = 0; i < FFT::elements_per_thread; i++) {
                // twisting
                acc_CUDA[index] = hipCmul(acc_CUDA[index], twiddleTable_CUDA[twist_idx]);
                thread_data[i] = complex_type {acc_CUDA[index].x, acc_CUDA[index].y};
                // FFT::stride shows how elements from a single FFT should be split between threads
                index += FFT::stride;
                twist_idx += FFT::stride;
            }
        }

        FFT().execute(thread_data, shared_mem);

        // Save results
        {
            unsigned int index = offset + threadIdx.x;
            for (unsigned i = 0; i < FFT::elements_per_thread; i++) {
                acc_CUDA[index] = make_hipDoubleComplex(thread_data[i].x, thread_data[i].y);
                // FFT::stride shows how elements from a single FFT should be split between threads
                index += FFT::stride;
            }
        }
    }
    else{ // must meet syncs made by FFT
        for(uint32_t i = 0; i < syncNum; ++i)
            __syncthreads();
    }
    __syncthreads();

    for(uint32_t round = 0; round < n; ++round){
        /* Copy acc_CUDA to ct_CUDA */
        for(uint32_t i = tid; i < N; i += bdim){
            ct_CUDA[i] = acc_CUDA[i];
        }
        __syncthreads();

        /* 2 times Inverse IFFT */
        if(threadIdx.y < 2){
            // Load data from shared memory to registers
            {
                unsigned int index = offset + threadIdx.x;
                for (unsigned i = 0; i < IFFT::elements_per_thread; i++) {
                    thread_data[i] = complex_type {ct_CUDA[index].x, ct_CUDA[index].y};
                    // FFT::stride shows how elements from a single FFT should be split between threads
                    index += IFFT::stride;
                }
            }

            // Scale values
            double scale = 1.0 / cufftdx::size_of<IFFT>::value;
            for (unsigned int i = 0; i < IFFT::elements_per_thread; i++) {
                thread_data[i].x *= scale;
                thread_data[i].y *= scale;
            }

            IFFT().execute(thread_data, shared_mem);
        
            // Save results
            {
                unsigned int index = offset + threadIdx.x;
                unsigned int twist_idx = threadIdx.x;
                for (unsigned i = 0; i < IFFT::elements_per_thread; i++) {
                    ct_CUDA[index].x = thread_data[i].x;
                    ct_CUDA[index].y = thread_data[i].y;
                    // twisting
                    ct_CUDA[index] = hipCmul(ct_CUDA[index], twiddleTable_CUDA[twist_idx + NHalf]);
                    // Round to INT128 and MOD
                    ct_CUDA[index].x = static_cast<double>(static_cast<__int128_t>(rint(ct_CUDA[index].x)) % static_cast<__int128_t>(Q));
                    if (ct_CUDA[index].x < 0)
                        ct_CUDA[index].x += static_cast<double>(Q);
                    if (ct_CUDA[index].x >= QHalf)
                        ct_CUDA[index].x -= static_cast<double>(Q);
                    ct_CUDA[index].y = static_cast<double>(static_cast<__int128_t>(rint(ct_CUDA[index].y)) % static_cast<__int128_t>(Q));
                    if (ct_CUDA[index].y < 0)
                        ct_CUDA[index].y += static_cast<double>(Q);
                    if (ct_CUDA[index].y >= QHalf)
                        ct_CUDA[index].y -= static_cast<double>(Q);
                    // IFFT::stride shows how elements from a single FFT should be split between threads
                    index += IFFT::stride;
                    twist_idx += IFFT::stride;
                }
            }
        }
        else{ // must meet syncs made by IFFT
            for(uint32_t i = 0; i < syncNum; ++i)
                __syncthreads();
        }
        __syncthreads();

        /* SignedDigitDecompose */
        // polynomial from a
        for (size_t k = tid; k < NHalf; k += bdim) {
            int64_t d0 = static_cast<int64_t>(ct_CUDA[k].x);
            int64_t d1 = static_cast<int64_t>(ct_CUDA[k].y);

            for (size_t l = 0; l < digitsG2; l += 2) {
                int64_t r0 = (d0 << gBitsMaxBits) >> gBitsMaxBits;
                d0 = (d0 - r0) >> gBits;
                if (r0 < 0)
                    r0 += static_cast<int64_t>(Q);
                if (r0 >= QHalf)
                    r0 -= static_cast<int64_t>(Q);
                dct_CUDA[l*NHalf + k].x = static_cast<double>(r0);

                int64_t r1 = (d1 << gBitsMaxBits) >> gBitsMaxBits;
                d1 = (d1 - r1) >> gBits;
                if (r1 < 0)
                    r1 += static_cast<int64_t>(Q);
                if (r1 >= QHalf)
                    r1 -= static_cast<int64_t>(Q);
                dct_CUDA[l*NHalf + k].y = static_cast<double>(r1);
            }
        }

        // polynomial from b
        for (size_t k = tid + NHalf; k < N; k += bdim) {
            int64_t d0 = static_cast<int64_t>(ct_CUDA[k].x);
            int64_t d1 = static_cast<int64_t>(ct_CUDA[k].y);

            for (size_t l = 0; l < digitsG2; l += 2) {
                int64_t r0 = (d0 << gBitsMaxBits) >> gBitsMaxBits;
                d0 = (d0 - r0) >> gBits;
                if (r0 < 0)
                    r0 += static_cast<int64_t>(Q);
                if (r0 >= QHalf)
                    r0 -= static_cast<int64_t>(Q);
                dct_CUDA[l*NHalf + k].x = static_cast<double>(r0);

                int64_t r1 = (d1 << gBitsMaxBits) >> gBitsMaxBits;
                d1 = (d1 - r1) >> gBits;
                if (r1 < 0)
                    r1 += static_cast<int64_t>(Q);
                if (r1 >= QHalf)
                    r1 -= static_cast<int64_t>(Q);
                dct_CUDA[l*NHalf + k].y = static_cast<double>(r1);
            }
        }
        __syncthreads();

        /* digitsG2 times Forward FFT */
        // Load data from shared memory to registers
        {
            unsigned int index = offset + threadIdx.x;
            unsigned int twist_idx = threadIdx.x;
            for (unsigned i = 0; i < FFT::elements_per_thread; i++) {
                // twisting
                dct_CUDA[index] = hipCmul(dct_CUDA[index], twiddleTable_CUDA[twist_idx]);
                thread_data[i] = complex_type {dct_CUDA[index].x, dct_CUDA[index].y};
                // FFT::stride shows how elements from a single FFT should be split between threads
                index += FFT::stride;
                twist_idx += FFT::stride;
            }
        }

        FFT().execute(thread_data, shared_mem);

        // Save results
        {
            unsigned int index = offset + threadIdx.x;
            for (unsigned i = 0; i < FFT::elements_per_thread; i++) {
                dct_CUDA[index] = make_hipDoubleComplex(thread_data[i].x, thread_data[i].y);
                // FFT::stride shows how elements from a single FFT should be split between threads
                index += FFT::stride;
            }
        }
        __syncthreads();

        /* Obtain monomial */
        // First obtain both monomial(index) for sk = 1 and monomial(-index) for sk = -1
        auto aNeg         = (M - a_CUDA[round]) % M;
        uint64_t indexPos = a_CUDA[round];
        uint64_t indexNeg = aNeg;
        // index is in range [0,m] - so we need to adjust the edge case when
        // index = m to index = 0
        if (indexPos == M)
            indexPos = 0;
        if (indexNeg == M)
            indexNeg = 0;
        
        /* ACC times Bootstrapping key and monomial */
        /* multiply with ek0 */
        // polynomial a
        for (uint32_t i = tid; i < NHalf; i += bdim){
            shared_mem[i] = complex_type(0.0, 0.0);
            for (uint32_t l = 0; l < digitsG2; ++l){
                shared_mem[i].x = fma(dct_CUDA[l*NHalf + i].x, GINX_bootstrappingKey_CUDA[round*RGSW_size + (l << 1)*NHalf + i].x, shared_mem[i].x);
                shared_mem[i].x = fma(-dct_CUDA[l*NHalf + i].y, GINX_bootstrappingKey_CUDA[round*RGSW_size + (l << 1)*NHalf + i].y, shared_mem[i].x);
                shared_mem[i].y = fma(dct_CUDA[l*NHalf + i].x, GINX_bootstrappingKey_CUDA[round*RGSW_size + (l << 1)*NHalf + i].y, shared_mem[i].y);
                shared_mem[i].y = fma(dct_CUDA[l*NHalf + i].y, GINX_bootstrappingKey_CUDA[round*RGSW_size + (l << 1)*NHalf + i].x, shared_mem[i].y);
            }
        }
        // polynomial b
        for (uint32_t i = tid; i < NHalf; i += bdim){
            shared_mem[NHalf + i] = complex_type(0.0, 0.0);
            for (uint32_t l = 0; l < digitsG2; ++l){
                shared_mem[NHalf + i].x = fma(dct_CUDA[l*NHalf + i].x, GINX_bootstrappingKey_CUDA[round*RGSW_size + ((l << 1) + 1)*NHalf + i].x, shared_mem[NHalf + i].x);
                shared_mem[NHalf + i].x = fma(-dct_CUDA[l*NHalf + i].y, GINX_bootstrappingKey_CUDA[round*RGSW_size + ((l << 1) + 1)*NHalf + i].y, shared_mem[NHalf + i].x);
                shared_mem[NHalf + i].y = fma(dct_CUDA[l*NHalf + i].x, GINX_bootstrappingKey_CUDA[round*RGSW_size + ((l << 1) + 1)*NHalf + i].y, shared_mem[NHalf + i].y);
                shared_mem[NHalf + i].y = fma(dct_CUDA[l*NHalf + i].y, GINX_bootstrappingKey_CUDA[round*RGSW_size + ((l << 1) + 1)*NHalf + i].x, shared_mem[NHalf + i].y);
            }
        }
        __syncthreads();
        /* multiply with postive monomial */
        // polynomial a
        for (uint32_t i = tid; i < NHalf; i += bdim){
            acc_CUDA[i].x = fma(shared_mem[i].x, monomial_CUDA[indexPos*NHalf + i].x, acc_CUDA[i].x);
            acc_CUDA[i].x = fma(-shared_mem[i].y, monomial_CUDA[indexPos*NHalf + i].y, acc_CUDA[i].x);
            acc_CUDA[i].y = fma(shared_mem[i].x, monomial_CUDA[indexPos*NHalf + i].y, acc_CUDA[i].y);
            acc_CUDA[i].y = fma(shared_mem[i].y, monomial_CUDA[indexPos*NHalf + i].x, acc_CUDA[i].y);
        }
        // polynomial b
        for (uint32_t i = tid; i < NHalf; i += bdim){
            acc_CUDA[NHalf + i].x = fma(shared_mem[NHalf + i].x, monomial_CUDA[indexPos*NHalf + i].x, acc_CUDA[NHalf + i].x);
            acc_CUDA[NHalf + i].x = fma(-shared_mem[NHalf + i].y, monomial_CUDA[indexPos*NHalf + i].y, acc_CUDA[NHalf + i].x);
            acc_CUDA[NHalf + i].y = fma(shared_mem[NHalf + i].x, monomial_CUDA[indexPos*NHalf + i].y, acc_CUDA[NHalf + i].y);
            acc_CUDA[NHalf + i].y = fma(shared_mem[NHalf + i].y, monomial_CUDA[indexPos*NHalf + i].x, acc_CUDA[NHalf + i].y);
        }        
        __syncthreads();

        /* multiply with ek1 */
        // polynomial a
        for (uint32_t i = tid; i < NHalf; i += bdim){
            shared_mem[i] = complex_type(0.0, 0.0);
            for (uint32_t l = 0; l < digitsG2; ++l){
                shared_mem[i].x = fma(dct_CUDA[l*NHalf + i].x, GINX_bootstrappingKey_CUDA[n*RGSW_size + round*RGSW_size + (l << 1)*NHalf + i].x, shared_mem[i].x);
                shared_mem[i].x = fma(-dct_CUDA[l*NHalf + i].y, GINX_bootstrappingKey_CUDA[n*RGSW_size + round*RGSW_size + (l << 1)*NHalf + i].y, shared_mem[i].x);
                shared_mem[i].y = fma(dct_CUDA[l*NHalf + i].x, GINX_bootstrappingKey_CUDA[n*RGSW_size + round*RGSW_size + (l << 1)*NHalf + i].y, shared_mem[i].y);
                shared_mem[i].y = fma(dct_CUDA[l*NHalf + i].y, GINX_bootstrappingKey_CUDA[n*RGSW_size + round*RGSW_size + (l << 1)*NHalf + i].x, shared_mem[i].y);
            }
        }
        // polynomial b
        for (uint32_t i = tid; i < NHalf; i += bdim){
            shared_mem[NHalf + i] = complex_type(0.0, 0.0);
            for (uint32_t l = 0; l < digitsG2; ++l){
                shared_mem[NHalf + i].x = fma(dct_CUDA[l*NHalf + i].x, GINX_bootstrappingKey_CUDA[n*RGSW_size + round*RGSW_size + ((l << 1) + 1)*NHalf + i].x, shared_mem[NHalf + i].x);
                shared_mem[NHalf + i].x = fma(-dct_CUDA[l*NHalf + i].y, GINX_bootstrappingKey_CUDA[n*RGSW_size + round*RGSW_size + ((l << 1) + 1)*NHalf + i].y, shared_mem[NHalf + i].x);
                shared_mem[NHalf + i].y = fma(dct_CUDA[l*NHalf + i].x, GINX_bootstrappingKey_CUDA[n*RGSW_size + round*RGSW_size + ((l << 1) + 1)*NHalf + i].y, shared_mem[NHalf + i].y);
                shared_mem[NHalf + i].y = fma(dct_CUDA[l*NHalf + i].y, GINX_bootstrappingKey_CUDA[n*RGSW_size + round*RGSW_size + ((l << 1) + 1)*NHalf + i].x, shared_mem[NHalf + i].y);
            }
        }
        __syncthreads();
        /* multiply with negative monomial */
        // polynomial a
        for (uint32_t i = tid; i < NHalf; i += bdim){
            acc_CUDA[i].x = fma(shared_mem[i].x, monomial_CUDA[indexNeg*NHalf + i].x, acc_CUDA[i].x);
            acc_CUDA[i].x = fma(-shared_mem[i].y, monomial_CUDA[indexNeg*NHalf + i].y, acc_CUDA[i].x);
            acc_CUDA[i].y = fma(shared_mem[i].x, monomial_CUDA[indexNeg*NHalf + i].y, acc_CUDA[i].y);
            acc_CUDA[i].y = fma(shared_mem[i].y, monomial_CUDA[indexNeg*NHalf + i].x, acc_CUDA[i].y);
        }
        // polynomial b
        for (uint32_t i = tid; i < NHalf; i += bdim){
            acc_CUDA[NHalf + i].x = fma(shared_mem[NHalf + i].x, monomial_CUDA[indexNeg*NHalf + i].x, acc_CUDA[NHalf + i].x);
            acc_CUDA[NHalf + i].x = fma(-shared_mem[NHalf + i].y, monomial_CUDA[indexNeg*NHalf + i].y, acc_CUDA[NHalf + i].x);
            acc_CUDA[NHalf + i].y = fma(shared_mem[NHalf + i].x, monomial_CUDA[indexNeg*NHalf + i].y, acc_CUDA[NHalf + i].y);
            acc_CUDA[NHalf + i].y = fma(shared_mem[NHalf + i].y, monomial_CUDA[indexNeg*NHalf + i].x, acc_CUDA[NHalf + i].y);
        }        
        __syncthreads();
    }

    /* 2 times Inverse IFFT */
    if(threadIdx.y < 2){
        // Load data from shared memory to registers
        {
            unsigned int index = offset + threadIdx.x;
            for (unsigned i = 0; i < IFFT::elements_per_thread; i++) {
                thread_data[i] = complex_type {acc_CUDA[index].x, acc_CUDA[index].y};
                // FFT::stride shows how elements from a single FFT should be split between threads
                index += IFFT::stride;
            }
        }

        // Scale values
        double scale = 1.0 / cufftdx::size_of<IFFT>::value;
        for (unsigned int i = 0; i < IFFT::elements_per_thread; i++) {
            thread_data[i].x *= scale;
            thread_data[i].y *= scale;
        }

        IFFT().execute(thread_data, shared_mem);
    
        // Save results
        {
            unsigned int index = offset + threadIdx.x;
            unsigned int twist_idx = threadIdx.x;
            for (unsigned i = 0; i < IFFT::elements_per_thread; i++) {
                acc_CUDA[index].x = thread_data[i].x;
                acc_CUDA[index].y = thread_data[i].y;
                // twisting
                acc_CUDA[index] = hipCmul(acc_CUDA[index], twiddleTable_CUDA[twist_idx + NHalf]);
                // Round to INT128 and MOD
                acc_CUDA[index].x = static_cast<double>(static_cast<__int128_t>(rint(acc_CUDA[index].x)) % static_cast<__int128_t>(Q));
                if (acc_CUDA[index].x < 0)
                    acc_CUDA[index].x += static_cast<double>(Q);
                acc_CUDA[index].y = static_cast<double>(static_cast<__int128_t>(rint(acc_CUDA[index].y)) % static_cast<__int128_t>(Q));
                if (acc_CUDA[index].y < 0)
                    acc_CUDA[index].y += static_cast<double>(Q);
                // IFFT::stride shows how elements from a single FFT should be split between threads
                index += IFFT::stride;
                twist_idx += FFT::stride;
            }
        }
    }
    else{ // must meet syncs made by IFFT
       for(uint32_t i = 0; i < syncNum; ++i)
            __syncthreads();
    }
    __syncthreads();

    /****************************************
    * the accumulator result is encrypted w.r.t. the transposed secret key
    * we can transpose "a" to get an encryption under the original secret key z
    * z = (z0, −zq/2−1, . . . , −z1)
    *****************************************/
    /* Copy acc_CUDA to ct_CUDA */
    for(uint32_t i = tid; i < NHalf; i += bdim){
        ct_CUDA[i] = acc_CUDA[i];
    }
    __syncthreads();

    for(uint32_t i = tid+1; i < NHalf; i += bdim){
        acc_CUDA[i].x = static_cast<double>((Q - static_cast<uint64_t>(ct_CUDA[NHalf - i].y)));
        acc_CUDA[i].y = static_cast<double>((Q - static_cast<uint64_t>(ct_CUDA[NHalf - i].x)));
    }
    if(tid == 0) acc_CUDA[0].y = static_cast<double>((Q - static_cast<uint64_t>(ct_CUDA[0].y)));
    __syncthreads();
}

template<class FFT>
__global__ void cuFFTDxFWD(Complex_d* data, Complex_d* twiddleTable_CUDA){
    /* cufftdx variables */
    using complex_type = typename FFT::value_type;
    const unsigned int local_fft_id = threadIdx.y;
    const unsigned int offset = cufftdx::size_of<FFT>::value * (blockIdx.x + local_fft_id);
    extern __shared__ complex_type shared_mem[];
    complex_type thread_data[FFT::storage_size];     
    
    // Load data from shared memory to registers
    {
        unsigned int index = offset + threadIdx.x;
        unsigned int twist_idx = threadIdx.x;
        for (unsigned i = 0; i < FFT::elements_per_thread; i++) {
            // twisting
            data[index] = hipCmul(data[index], twiddleTable_CUDA[twist_idx]);
            thread_data[i] = complex_type {data[index].x, data[index].y};
            // FFT::stride shows how elements from a single FFT should be split between threads
            index += FFT::stride;
            twist_idx += FFT::stride;
        }
    }

    FFT().execute(thread_data, shared_mem);

    // Save results
    {
        unsigned int index = offset + threadIdx.x;
        for (unsigned i = 0; i < FFT::elements_per_thread; i++) {
            data[index] = make_hipDoubleComplex(thread_data[i].x, thread_data[i].y);
            // FFT::stride shows how elements from a single FFT should be split between threads
            index += FFT::stride;
        }
    }
}

void GPUSetup(std::shared_ptr<std::vector<std::vector<std::vector<std::shared_ptr<std::vector<std::vector<std::vector<Complex>>>>>>>> GINX_bootstrappingKey_FFT, 
    const std::shared_ptr<RingGSWCryptoParams> RGSWParams, LWESwitchingKey keySwitchingKey, const std::shared_ptr<LWECryptoParams> LWEParams)
{
    std::cout << "GPU Setup Start\n";

    /* Setting up available GPU INFO */
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cerr << "No CUDA devices found." << std::endl;
        return;
    }

    for (int device = 0; device < deviceCount; ++device) {
        hipSetDevice(device);
        hipDeviceProp_t deviceProperties;
        hipGetDeviceProperties(&deviceProperties, device);

        GPUInfo info;
        info.name = deviceProperties.name;
        info.major = deviceProperties.major;
        info.minor = deviceProperties.minor;
        info.sharedMemoryPerBlock = deviceProperties.sharedMemPerBlock;
        info.maxBlocksPerMultiprocessor = deviceProperties.maxThreadsPerMultiProcessor / deviceProperties.maxThreadsPerBlock;
        info.maxThreadsPerBlock = deviceProperties.maxThreadsPerBlock;
        info.maxGridX = deviceProperties.maxGridSize[0];
        info.maxGridY = deviceProperties.maxGridSize[1];
        info.maxGridZ = deviceProperties.maxGridSize[2];
        info.maxBlockX = deviceProperties.maxThreadsDim[0];
        info.maxBlockY = deviceProperties.maxThreadsDim[1];
        info.maxBlockZ = deviceProperties.maxThreadsDim[2];
        info.warpSize = deviceProperties.warpSize;
        info.multiprocessorCount = deviceProperties.multiProcessorCount;

        gpuInfoList.push_back(info);
    }

    /* Parameters Set */
    uint32_t N          = RGSWParams->GetN();
    uint32_t NHalf      = N >> 1;
    uint32_t digitsG2   = RGSWParams->GetDigitsG() << 1;
    uint32_t arch       = gpuInfoList[0].major * 100 + gpuInfoList[0].minor * 10;

    /* Determine template of GPUSetup_core */
    switch (arch){
        case 700: // V100
            switch (NHalf){
                case 512:
                    switch (digitsG2){
                        case 2:
                            GPUSetup_core<700, 512, 2>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 4:
                            GPUSetup_core<700, 512, 4>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 6:
                            GPUSetup_core<700, 512, 6>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 8:
                            GPUSetup_core<700, 512, 8>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 10:
                            GPUSetup_core<700, 512, 10>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 12:
                            GPUSetup_core<700, 512, 12>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 1024:
                    switch (digitsG2){
                        case 2:
                            GPUSetup_core<700, 1024, 2>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 4:
                            GPUSetup_core<700, 1024, 4>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 6:
                            GPUSetup_core<700, 1024, 6>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 8:
                            GPUSetup_core<700, 1024, 8>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 10:
                            GPUSetup_core<700, 1024, 10>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 12:
                            GPUSetup_core<700, 1024, 12>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 2048:
                    switch (digitsG2){
                        case 2:
                            GPUSetup_core<700, 2048, 2>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 4:
                            GPUSetup_core<700, 2048, 4>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 6:
                            GPUSetup_core<700, 2048, 6>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 8:
                            GPUSetup_core<700, 2048, 8>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 10:
                            GPUSetup_core<700, 2048, 10>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 12:
                            GPUSetup_core<700, 2048, 12>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                default:
                    std::cerr << "Unsupported N\n";
                    exit(1);
            }
            break;
        case 800: // A100
            switch (NHalf){
                case 512:
                    switch (digitsG2){
                        case 2:
                            GPUSetup_core<800, 512, 2>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 4:
                            GPUSetup_core<800, 512, 4>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 6:
                            GPUSetup_core<800, 512, 6>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 8:
                            GPUSetup_core<800, 512, 8>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 10:
                            GPUSetup_core<800, 512, 10>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 12:
                            GPUSetup_core<800, 512, 12>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 1024:
                    switch (digitsG2){
                        case 2:
                            GPUSetup_core<800, 1024, 2>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 4:
                            GPUSetup_core<800, 1024, 4>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 6:
                            GPUSetup_core<800, 1024, 6>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 8:
                            GPUSetup_core<800, 1024, 8>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 10:
                            GPUSetup_core<800, 1024, 10>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 12:
                            GPUSetup_core<800, 1024, 12>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 2048:
                    switch (digitsG2){
                        case 2:
                            GPUSetup_core<800, 2048, 2>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 4:
                            GPUSetup_core<800, 2048, 4>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 6:
                            GPUSetup_core<800, 2048, 6>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 8:
                            GPUSetup_core<800, 2048, 8>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 10:
                            GPUSetup_core<800, 2048, 10>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 12:
                            GPUSetup_core<800, 2048, 12>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                default:
                    std::cerr << "Unsupported N\n";
                    exit(1);
            }
            break;
        case 860: // RTX30 series
            switch (NHalf){
                case 512:
                    switch (digitsG2){
                        case 2:
                            GPUSetup_core<860, 512, 2>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 4:
                            GPUSetup_core<860, 512, 4>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 6:
                            GPUSetup_core<860, 512, 6>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 8:
                            GPUSetup_core<860, 512, 8>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 10:
                            GPUSetup_core<860, 512, 10>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 12:
                            GPUSetup_core<860, 512, 12>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 1024:
                    switch (digitsG2){
                        case 2:
                            GPUSetup_core<860, 1024, 2>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 4:
                            GPUSetup_core<860, 1024, 4>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 6:
                            GPUSetup_core<860, 1024, 6>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 8:
                            GPUSetup_core<860, 1024, 8>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 10:
                            GPUSetup_core<860, 1024, 10>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 12:
                            GPUSetup_core<860, 1024, 12>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 2048:
                    switch (digitsG2){
                        case 2:
                            GPUSetup_core<860, 2048, 2>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 4:
                            GPUSetup_core<860, 2048, 4>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 6:
                            GPUSetup_core<860, 2048, 6>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 8:
                            GPUSetup_core<860, 2048, 8>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 10:
                            GPUSetup_core<860, 2048, 10>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 12:
                            GPUSetup_core<860, 2048, 12>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                default:
                    std::cerr << "Unsupported N\n";
                    exit(1);
            }
            break;
        case 890: // RTX40 series
            switch (NHalf){
                case 512:
                    switch (digitsG2){
                        case 2:
                            GPUSetup_core<890, 512, 2>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 4:
                            GPUSetup_core<890, 512, 4>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 6:
                            GPUSetup_core<890, 512, 6>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 8:
                            GPUSetup_core<890, 512, 8>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 10:
                            GPUSetup_core<890, 512, 10>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 12:
                            GPUSetup_core<890, 512, 12>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 1024:
                    switch (digitsG2){
                        case 2:
                            GPUSetup_core<890, 1024, 2>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 4:
                            GPUSetup_core<890, 1024, 4>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 6:
                            GPUSetup_core<890, 1024, 6>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 8:
                            GPUSetup_core<890, 1024, 8>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 10:
                            GPUSetup_core<890, 1024, 10>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 12:
                            GPUSetup_core<890, 1024, 12>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 2048:
                    switch (digitsG2){
                        case 2:
                            GPUSetup_core<890, 2048, 2>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 4:
                            GPUSetup_core<890, 2048, 4>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 6:
                            GPUSetup_core<890, 2048, 6>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 8:
                            GPUSetup_core<890, 2048, 8>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 10:
                            GPUSetup_core<890, 2048, 10>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 12:
                            GPUSetup_core<890, 2048, 12>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                default:
                    std::cerr << "Unsupported N\n";
                    exit(1);
            }
            break;
        case 900: // H100
            switch (NHalf){
                case 512:
                    switch (digitsG2){
                        case 2:
                            GPUSetup_core<900, 512, 2>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 4:
                            GPUSetup_core<900, 512, 4>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 6:
                            GPUSetup_core<900, 512, 6>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 8:
                            GPUSetup_core<900, 512, 8>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 10:
                            GPUSetup_core<900, 512, 10>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 12:
                            GPUSetup_core<900, 512, 12>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 1024:
                    switch (digitsG2){
                        case 2:
                            GPUSetup_core<900, 1024, 2>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 4:
                            GPUSetup_core<900, 1024, 4>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 6:
                            GPUSetup_core<900, 1024, 6>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 8:
                            GPUSetup_core<900, 1024, 8>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 10:
                            GPUSetup_core<900, 1024, 10>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 12:
                            GPUSetup_core<900, 1024, 12>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 2048:
                    switch (digitsG2){
                        case 2:
                            GPUSetup_core<900, 2048, 2>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 4:
                            GPUSetup_core<900, 2048, 4>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 6:
                            GPUSetup_core<900, 2048, 6>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 8:
                            GPUSetup_core<900, 2048, 8>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 10:
                            GPUSetup_core<900, 2048, 10>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        case 12:
                            GPUSetup_core<900, 2048, 12>(GINX_bootstrappingKey_FFT, RGSWParams, keySwitchingKey, LWEParams);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                default:
                    std::cerr << "Unsupported N\n";
                    exit(1);
            }
            break;
        default:
            std::cerr << "Unsupported GPU architecture\n";
            exit(1);
    }
    
    std::cout << "GPU Setup Done\n";
}

template<uint32_t arch, uint32_t FFT_dimension, uint32_t FFT_num>
void GPUSetup_core(std::shared_ptr<std::vector<std::vector<std::vector<std::shared_ptr<std::vector<std::vector<std::vector<Complex>>>>>>>> GINX_bootstrappingKey_FFT, 
    const std::shared_ptr<RingGSWCryptoParams> RGSWParams, LWESwitchingKey keySwitchingKey, const std::shared_ptr<LWECryptoParams> LWEParams)
{
    /* Parameters Set */
    auto Q            = RGSWParams->GetQ();
    NativeInteger QHalf = Q >> 1;
    NativeInteger::SignedNativeInt Q_int = Q.ConvertToInt();
    uint32_t N            = RGSWParams->GetN();
    uint32_t NHalf     = N >> 1;
    uint32_t n = (*GINX_bootstrappingKey_FFT)[0][0].size();
    uint32_t digitsG2 = RGSWParams->GetDigitsG() << 1;
    uint32_t baseG = RGSWParams->GetBaseG();
    uint32_t RGSW_size = digitsG2 * 2 * NHalf;
    NativeInteger qKS = LWEParams->GetqKS();
    uint32_t baseKS   = LWEParams->GetBaseKS();
    uint32_t digitCountKS = (uint32_t)std::ceil(log(qKS.ConvertToDouble()) / log(static_cast<double>(baseKS)));

    int SM_count = gpuInfoList[0].multiprocessorCount;

    /* Create cuda streams */
    streams.resize(gpuInfoList[0].multiprocessorCount);
    for (int s = 0; s < gpuInfoList[0].multiprocessorCount; s++) {
        hipStreamCreate(&streams[s]);
    }

    /* Configure cuFFTDx */
    using FFT     = decltype(cufftdx::Block() + cufftdx::Size<FFT_dimension>() + cufftdx::Type<cufftdx::fft_type::c2c>() + cufftdx::Direction<cufftdx::fft_direction::forward>() + cufftdx::ElementsPerThread<8>() +
                        cufftdx::Precision<double>() + cufftdx::FFTsPerBlock<FFT_num>() + cufftdx::SM<arch>());

    using IFFT     = decltype(cufftdx::Block() + cufftdx::Size<FFT_dimension>() + cufftdx::Type<cufftdx::fft_type::c2c>() + cufftdx::Direction<cufftdx::fft_direction::inverse>() + cufftdx::ElementsPerThread<8>() +
                            cufftdx::Precision<double>() + cufftdx::FFTsPerBlock<2>() + cufftdx::SM<arch>());

    using FFT_multi      = decltype(cufftdx::Block() + cufftdx::Size<FFT_dimension>() + cufftdx::Type<cufftdx::fft_type::c2c>() + cufftdx::Direction<cufftdx::fft_direction::forward>() +
                            cufftdx::Precision<double>() + cufftdx::FFTsPerBlock<2>() + cufftdx::SM<arch>());

    using IFFT_multi     = decltype(cufftdx::Block() + cufftdx::Size<FFT_dimension>() + cufftdx::Type<cufftdx::fft_type::c2c>() + cufftdx::Direction<cufftdx::fft_direction::inverse>() +
                            cufftdx::Precision<double>() + cufftdx::FFTsPerBlock<2>() + cufftdx::SM<arch>());

    using FFT_fwd  = decltype(cufftdx::Block() + cufftdx::Size<FFT_dimension>() + cufftdx::Type<cufftdx::fft_type::c2c>() + cufftdx::Direction<cufftdx::fft_direction::forward>() + cufftdx::ElementsPerThread<8>() +
                        cufftdx::Precision<double>() + cufftdx::FFTsPerBlock<1>() + cufftdx::SM<arch>());

    /* Increase max shared memory */
    // Single block Bootstrapping shared memory size
    if(FFT::shared_memory_size > 65536)
        hipFuncSetAttribute(reinterpret_cast<const void*>(bootstrappingSingleBlock<FFT), IFFT>, hipFuncAttributePreferredSharedMemoryCarveout, 100);
    else if(FFT::shared_memory_size > 32768)
        hipFuncSetAttribute(reinterpret_cast<const void*>(bootstrappingSingleBlock<FFT), IFFT>, hipFuncAttributePreferredSharedMemoryCarveout, 64);
    else
        hipFuncSetAttribute(reinterpret_cast<const void*>(bootstrappingSingleBlock<FFT), IFFT>, hipFuncAttributePreferredSharedMemoryCarveout, 32);
    hipFuncSetAttribute(reinterpret_cast<const void*>(bootstrappingSingleBlock<FFT), IFFT>, hipFuncAttributeMaxDynamicSharedMemorySize, FFT::shared_memory_size);

    // Multi block Bootstrapping shared memory size
    if(FFT_multi::shared_memory_size > 65536)
        hipFuncSetAttribute(reinterpret_cast<const void*>(bootstrappingMultiBlock<FFT_multi), IFFT_multi>, hipFuncAttributePreferredSharedMemoryCarveout, 100);
    else if(FFT_multi::shared_memory_size > 32768)
        hipFuncSetAttribute(reinterpret_cast<const void*>(bootstrappingMultiBlock<FFT_multi), IFFT_multi>, hipFuncAttributePreferredSharedMemoryCarveout, 64);
    else
        hipFuncSetAttribute(reinterpret_cast<const void*>(bootstrappingMultiBlock<FFT_multi), IFFT_multi>, hipFuncAttributePreferredSharedMemoryCarveout, 32);
    hipFuncSetAttribute(reinterpret_cast<const void*>(bootstrappingMultiBlock<FFT_multi), IFFT_multi>, hipFuncAttributeMaxDynamicSharedMemorySize, FFT_multi::shared_memory_size);

    // MKMSwitch shared memory size
    hipFuncSetAttribute(reinterpret_cast<const void*>(MKMSwitchKernel), hipFuncAttributeMaxDynamicSharedMemorySize, (n + 1) * sizeof(uint64_t));

    // cuFFTDx Forward shared memory size
    hipFuncSetAttribute(reinterpret_cast<const void*>(cuFFTDxFWD<FFT_fwd>), hipFuncAttributePreferredSharedMemoryCarveout, 64);
    hipFuncSetAttribute(reinterpret_cast<const void*>(cuFFTDxFWD<FFT_fwd>), hipFuncAttributeMaxDynamicSharedMemorySize, FFT_fwd::shared_memory_size);

    /* Initialize twiddle table */
    Complex *twiddleTable;
    hipHostMalloc((void**)&twiddleTable, 2 * NHalf * sizeof(Complex));
    for (size_t j = 0; j < NHalf; j++) {
        twiddleTable[j] = Complex(cos(static_cast<double>(2 * M_PI * j)/ (N << 1)), sin(static_cast<double>(2 * M_PI * j) / (N << 1)));
    }
    for (size_t j = NHalf; j < N; j++) {
        twiddleTable[j] = Complex(cos(static_cast<double>(-2 * M_PI * (j - NHalf)) / (N << 1)), sin(static_cast<double>(-2 * M_PI * (j - NHalf)) / (N << 1)));
    }
    // Bring twiddle table to GPU
    hipMalloc(&twiddleTable_CUDA, 2 * NHalf * sizeof(Complex_d));
    hipMemcpy(twiddleTable_CUDA, twiddleTable, 2 * NHalf * sizeof(Complex_d), hipMemcpyHostToDevice);
    hipHostFree(twiddleTable);

    /* Initialize params_CUDA */
    uint64_t *paramters;
    hipHostMalloc((void**)&paramters, 6 * sizeof(uint64_t));
    paramters[0] = N;
    paramters[1] = n;
    paramters[2] = static_cast<uint64_t>(Q_int);
    paramters[3] = digitsG2;
    paramters[4] = baseG;
    auto it = synchronizationMap.find({arch, FFT_dimension});
    if (it != synchronizationMap.end() && it->second != 0) {
        paramters[5] = static_cast<uint64_t>(it->second);
    } else {
        std::cerr << "Hasn't tested on this GPU yet, please contact r11922138@ntu.edu.tw" << std::endl;
        exit(1);
    }
    // Bring params_CUDA to GPU
    hipMalloc(&params_CUDA, 6 * sizeof(uint64_t));
    hipMemcpy(params_CUDA, paramters, 6 * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipHostFree(paramters);

    /* Initialize bootstrapping key */
    Complex *bootstrappingKey;
    hipHostMalloc((void**)&bootstrappingKey, 2 * n * RGSW_size * sizeof(Complex)); // ternery needs two secret keys
    for(int num_key = 0; num_key < 2; num_key++){
        for(int i = 0; i < n; i++){
            for(int l = 0; l < digitsG2; l++){
                for(int m = 0; m < 2; m++){
                    std::vector<Complex> temp = (*(*GINX_bootstrappingKey_FFT)[0][num_key][i])[l][m];
                    DiscreteFourierTransform::NegacyclicInverseTransform(temp);
                    for(int j = 0; j < NHalf; j++){
                        bootstrappingKey[num_key*n*RGSW_size + i*RGSW_size + l*2*NHalf + m*NHalf + j] = Complex(temp[j].real(), temp[j + NHalf].real());
                    }
                }
            }
        }
    }
    // Bring bootstrapping key to GPU
    hipMalloc(&GINX_bootstrappingKey_CUDA, 2 * n * RGSW_size * sizeof(Complex_d));
    hipMemcpy(GINX_bootstrappingKey_CUDA, bootstrappingKey, 2 * n * RGSW_size * sizeof(Complex_d), hipMemcpyHostToDevice);
    hipHostFree(bootstrappingKey);
    cuFFTDxFWD<FFT_fwd><<<2 * n * digitsG2 * 2, FFT_fwd::block_dim, FFT_fwd::shared_memory_size>>>(GINX_bootstrappingKey_CUDA, twiddleTable_CUDA);
    hipDeviceSynchronize();

    /* Initialize keySwitching key */
    uint64_t *keySwitchingkey_host;
    hipHostMalloc((void**)&keySwitchingkey_host, N * baseKS * digitCountKS * (n + 1) * sizeof(uint64_t));
    // A
    for(int i = 0; i < N; i++){
        for(int j = 0; j < baseKS; j++){
            for(int k = 0; k < digitCountKS; k++){
                for(int l = 0; l < n; l++){
                    keySwitchingkey_host[i*baseKS*digitCountKS*n + j*digitCountKS*n + k*n + l] 
                        = static_cast<uint64_t>(keySwitchingKey->GetElementsA()[i][j][k][l].ConvertToInt());
                }
            }
        }
    }
    // B
    for(int i = 0; i < N; i++){
        for(int j = 0; j < baseKS; j++){
            for(int k = 0; k < digitCountKS; k++){
                keySwitchingkey_host[N*baseKS*digitCountKS*n + i*baseKS*digitCountKS + j*digitCountKS + k] 
                    = static_cast<uint64_t>(keySwitchingKey->GetElementsB()[i][j][k].ConvertToInt());
            }
        }
    }
    // Bring keySwitching key to GPU
    hipMalloc(&keySwitchingkey_CUDA, N * baseKS * digitCountKS * (n + 1) * sizeof(uint64_t));
    hipMemcpy(keySwitchingkey_CUDA, keySwitchingkey_host, N * baseKS * digitCountKS * (n + 1) * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipHostFree(keySwitchingkey_host);

    /* Initialize monomial array */
    Complex *monomial_arr;
    hipHostMalloc((void**)&monomial_arr, 2 * N * NHalf * sizeof(Complex));
    // loop for positive values of m
    std::vector<Complex> monomial(N, Complex(0.0, 0.0));
    for (size_t m_count = 0; m_count < N; ++m_count) {
        NativePoly monomial_t    = RGSWParams->GetMonomial(m_count);
        monomial_t.SetFormat(Format::COEFFICIENT);
        for (size_t i = 0; i < N; ++i) {
            NativeInteger::SignedNativeInt d = (monomial_t[i] < QHalf) ? monomial_t[i].ConvertToInt() : (monomial_t[i].ConvertToInt() - Q_int);
            monomial[i] = Complex (static_cast<double>(d), 0);
        }
        for (size_t i = 0; i < NHalf; ++i) 
            monomial_arr[m_count*NHalf + i] = Complex(monomial[i].real(), monomial[i + NHalf].real());
    }
    // loop for negative values of m
    std::vector<Complex> monomialNeg(N, Complex(0.0, 0.0));
    for (size_t m_count = N; m_count < (N << 1); ++m_count) {   
        NativePoly monomialNeg_t = RGSWParams->GetMonomial(m_count);
        monomialNeg_t.SetFormat(Format::COEFFICIENT);
        for (size_t i = 0; i < N; ++i) {
            NativeInteger::SignedNativeInt d = (monomialNeg_t[i] < QHalf) ? monomialNeg_t[i].ConvertToInt() : (monomialNeg_t[i].ConvertToInt() - Q_int);
            monomialNeg[i] = Complex (static_cast<double>(d), 0);
        }
        for (size_t i = 0; i < NHalf; ++i) 
            monomial_arr[m_count*NHalf + i] = Complex(monomialNeg[i].real(), monomialNeg[i + NHalf].real());
    }
    // Bring monomial array to GPU
    hipMalloc(&monomial_CUDA, 2 * N * NHalf * sizeof(Complex_d));
    hipMemcpy(monomial_CUDA, monomial_arr, 2 * N * NHalf * sizeof(Complex_d), hipMemcpyHostToDevice);
    hipHostFree(monomial_arr);
    cuFFTDxFWD<FFT_fwd><<<2 * N, FFT_fwd::block_dim, FFT_fwd::shared_memory_size>>>(monomial_CUDA, twiddleTable_CUDA);
    hipDeviceSynchronize();

    /* Allocate ct_CUDA on GPU */
    hipMalloc(&ct_CUDA, SM_count * 2 * NHalf * sizeof(Complex_d));

    /* Allocate dct_CUDA on GPU */
    hipMalloc(&dct_CUDA, SM_count * digitsG2 * NHalf * sizeof(Complex_d));

    /* Allocate acc_CUDA on GPU */
    hipMalloc(&acc_CUDA, SM_count * 2 * NHalf * sizeof(Complex_d));

    /* Allocate a_CUDA on GPU */
    hipMalloc(&a_CUDA, SM_count * n * sizeof(uint64_t));
}

void AddToAccCGGI_CUDA(const std::shared_ptr<RingGSWCryptoParams> params, const NativeVector& a, std::vector<std::vector<Complex>>& acc_d, std::string mode)
{   
    /* Parameters Set */
    uint32_t N            = params->GetN();
    uint32_t NHalf     = N >> 1;
    uint32_t digitsG2 = params->GetDigitsG() << 1;
    uint32_t arch = gpuInfoList[0].major * 100 + gpuInfoList[0].minor * 10;

    /* Determine template of AddToAccCGGI_CUDA_core */
    switch (arch){
        case 700: // V100
            switch (NHalf){
                case 512:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<700, 512, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<700, 512, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<700, 512, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<700, 512, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<700, 512, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<700, 512, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 1024:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<700, 1024, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<700, 1024, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<700, 1024, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<700, 1024, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<700, 1024, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<700, 1024, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 2048:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<700, 2048, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<700, 2048, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<700, 2048, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<700, 2048, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<700, 2048, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<700, 2048, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                default:
                    std::cerr << "Unsupported N\n";
                    exit(1);
            }
            break;
        case 800: // A100
            switch (NHalf){
                case 512:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<800, 512, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<800, 512, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<800, 512, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<800, 512, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<800, 512, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<800, 512, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 1024:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<800, 1024, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<800, 1024, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<800, 1024, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<800, 1024, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<800, 1024, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<800, 1024, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 2048:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<800, 2048, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<800, 2048, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<800, 2048, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<800, 2048, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<800, 2048, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<800, 2048, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                default:
                    std::cerr << "Unsupported N\n";
                    exit(1);
            }
            break;
        case 860: // RTX30 series
            switch (NHalf){
                case 512:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<860, 512, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<860, 512, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<860, 512, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<860, 512, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<860, 512, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<860, 512, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 1024:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<860, 1024, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<860, 1024, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<860, 1024, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<860, 1024, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<860, 1024, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<860, 1024, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 2048:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<860, 2048, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<860, 2048, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<860, 2048, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<860, 2048, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<860, 2048, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<860, 2048, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                default:
                    std::cerr << "Unsupported N\n";
                    exit(1);
            }
            break;
        case 890: // RTX40 series
            switch (NHalf){
                case 512:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<890, 512, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<890, 512, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<890, 512, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<890, 512, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<890, 512, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<890, 512, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 1024:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<890, 1024, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<890, 1024, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<890, 1024, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<890, 1024, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<890, 1024, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<890, 1024, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 2048:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<890, 2048, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<890, 2048, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<890, 2048, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<890, 2048, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<890, 2048, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<890, 2048, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                default:
                    std::cerr << "Unsupported N\n";
                    exit(1);
            }
            break;
        case 900: // H100
            switch (NHalf){
                case 512:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<900, 512, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<900, 512, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<900, 512, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<900, 512, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<900, 512, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<900, 512, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 1024:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<900, 1024, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<900, 1024, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<900, 1024, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<900, 1024, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<900, 1024, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<900, 1024, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 2048:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<900, 2048, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<900, 2048, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<900, 2048, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<900, 2048, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<900, 2048, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<900, 2048, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                default:
                    std::cerr << "Unsupported N\n";
                    exit(1);
            }
            break;
        default:
            std::cerr << "Unsupported GPU architecture\n";
            exit(1);
    }
}

template<uint32_t arch, uint32_t FFT_dimension, uint32_t FFT_num>
void AddToAccCGGI_CUDA_core(const std::shared_ptr<RingGSWCryptoParams> params, const NativeVector& a, std::vector<std::vector<Complex>>& acc_d, std::string mode)
{   
    /* parameters set */
    auto mod        = a.GetModulus();
    uint32_t modInt = mod.ConvertToInt();
    auto Q            = params->GetQ();
    NativeInteger QHalf = Q >> 1;
    NativeInteger::SignedNativeInt Q_int = Q.ConvertToInt();
    uint32_t N         = params->GetN();
    uint32_t NHalf     = N >> 1;
    uint32_t n =  a.GetLength();
    uint32_t M      = 2 * params->GetN();
    uint32_t digitsG2 = params->GetDigitsG() << 1;

    /* Configure cuFFTDx */
    using FFT     = decltype(cufftdx::Block() + cufftdx::Size<FFT_dimension>() + cufftdx::Type<cufftdx::fft_type::c2c>() + cufftdx::Direction<cufftdx::fft_direction::forward>() + cufftdx::ElementsPerThread<8>() +
                        cufftdx::Precision<double>() + cufftdx::FFTsPerBlock<FFT_num>() + cufftdx::SM<arch>());

    using IFFT     = decltype(cufftdx::Block() + cufftdx::Size<FFT_dimension>() + cufftdx::Type<cufftdx::fft_type::c2c>() + cufftdx::Direction<cufftdx::fft_direction::inverse>() + cufftdx::ElementsPerThread<8>() +
                            cufftdx::Precision<double>() + cufftdx::FFTsPerBlock<2>() + cufftdx::SM<arch>());

    using FFT_multi      = decltype(cufftdx::Block() + cufftdx::Size<FFT_dimension>() + cufftdx::Type<cufftdx::fft_type::c2c>() + cufftdx::Direction<cufftdx::fft_direction::forward>() +
                            cufftdx::Precision<double>() + cufftdx::FFTsPerBlock<2>() + cufftdx::SM<arch>());

    using IFFT_multi     = decltype(cufftdx::Block() + cufftdx::Size<FFT_dimension>() + cufftdx::Type<cufftdx::fft_type::c2c>() + cufftdx::Direction<cufftdx::fft_direction::inverse>() +
                            cufftdx::Precision<double>() + cufftdx::FFTsPerBlock<2>() + cufftdx::SM<arch>());

    /* Check whether block size exceeds cuda limitation */
    if(mode == "SINGLE"){
        if((NHalf / FFT::elements_per_thread * digitsG2) > gpuInfoList[0].maxThreadsPerBlock){
            std::cerr << "Exceed Maximum blocks per threads (" << gpuInfoList[0].maxThreadsPerBlock << ")\n";
            std::cerr << "Using " << (NHalf / FFT::elements_per_thread * digitsG2) << " threads" << ")\n";
            std::cerr << "NHalf: " << NHalf << "FFT::elements_per_thread: " << FFT::elements_per_thread << "digitsG2: " << digitsG2 << ")\n";
            exit(1);
        }
    }
    else if(mode == "MULTI"){
        if((NHalf / FFT_multi::elements_per_thread * 2) > gpuInfoList[0].maxThreadsPerBlock){
            std::cerr << "Exceed Maximum blocks per threads (" << gpuInfoList[0].maxThreadsPerBlock << ")\n";
            std::cerr << "Using " << (NHalf / FFT_multi::elements_per_thread * digitsG2) << " threads" << ")\n";
            std::cerr << "NHalf: " << NHalf << "FFT::elements_per_thread: " << FFT_multi::elements_per_thread << ")\n";
            exit(1);
        }
    }

    /* Check whether shared memory size exceeds cuda limitation */
    if(mode == "SINGLE"){
        if(FFT::shared_memory_size > gpuInfoList[0].sharedMemoryPerBlock){
            std::cerr << "Exceed Maximum sharedMemoryPerBlock ("<< gpuInfoList[0].sharedMemoryPerBlock << ")\n";
            std::cerr << "Declare "<< FFT::shared_memory_size << " now" << "\n";
            exit(1);
        }
    }
    else if(mode == "MULTI"){
        if(FFT_multi::shared_memory_size > gpuInfoList[0].sharedMemoryPerBlock){
            std::cerr << "Exceed Maximum sharedMemoryPerBlock ("<< gpuInfoList[0].sharedMemoryPerBlock << ")\n";
            std::cerr << "Declare "<< FFT_multi::shared_memory_size << " now" << "\n";
            exit(1);
        }
    }

    /* Initialize a_arr */
    uint64_t* a_arr;
    hipHostMalloc((void**)&a_arr, n * sizeof(uint64_t));
    for (size_t i = 0; i < n; ++i) {
        a_arr[i] = (mod.ModSub(a[i], mod) * (M / modInt)).ConvertToInt();
    }
    // Bring a to GPU
    hipMemcpy(a_CUDA, a_arr, n * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipHostFree(a_arr);

    /* Initialize acc_d_arr */
    Complex* acc_d_arr;
    hipHostMalloc((void**)&acc_d_arr, 2 * NHalf * sizeof(Complex));
    for(int i = 0; i < 2; i++)
        for(int j = 0; j < NHalf; j++)
            acc_d_arr[i*NHalf + j] = Complex(acc_d[i][j].real(), acc_d[i][j + NHalf].real());   
    // Bring acc_d to GPU
    hipMemcpy(acc_CUDA, acc_d_arr, 2 * NHalf * sizeof(Complex_d), hipMemcpyHostToDevice);

    /* Launch boostrapping kernel */
    if(mode == "SINGLE"){
        bootstrappingSingleBlock<FFT, IFFT><<<1, FFT::block_dim, FFT::shared_memory_size>>>
            (acc_CUDA, ct_CUDA, dct_CUDA, a_CUDA, monomial_CUDA, twiddleTable_CUDA, params_CUDA, GINX_bootstrappingKey_CUDA);
    }
    else if(mode == "MULTI"){
        void *kernelArgs[] = {(void *)&acc_CUDA, (void *)&ct_CUDA, (void *)&dct_CUDA, (void *)&a_CUDA, 
            (void *)&monomial_CUDA, (void *)&twiddleTable_CUDA, (void *)&params_CUDA, (void *)&GINX_bootstrappingKey_CUDA};
        hipLaunchCooperativeKernel((void*)(bootstrappingMultiBlock<FFT_multi, IFFT_multi>), digitsG2/2, FFT_multi::block_dim, kernelArgs, FFT_multi::shared_memory_size);
    }
    CUDA_CHECK_AND_EXIT(hipPeekAtLastError());
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    /* Copy acc_d_arr back to acc_d */
    hipMemcpy(acc_d_arr, acc_CUDA, 2 * NHalf * sizeof(Complex_d), hipMemcpyDeviceToHost);
    for(int i = 0; i < 2; i++){
        for(int j = 0; j < NHalf; j++){
            acc_d[i][j] = Complex(acc_d_arr[i*NHalf + j].real(), 0);
            acc_d[i][j + NHalf] = Complex(acc_d_arr[i*NHalf + j].imag(), 0);
        }
    }

    // /* Debugging txt files */
    // std::ofstream outputFile;
    // outputFile.open("acc.txt", std::ios::out);
    // for(uint32_t i = 0; i < 2; i++)
    //     for(uint32_t j = 0; j < (N >> 1); j++)
    //         outputFile << "(" << acc_d_arr[i*(N >> 1) + j].real() << ", " << acc_d_arr[i*(N >> 1) + j].imag() << ")" << std::endl;
    // outputFile.close();

    // // Copy the ct_CUDA back to the host
    // Complex* ct_arr;
    // hipHostMalloc((void**)&ct_arr, 2 * NHalf * sizeof(Complex));
    // hipMemcpy(ct_arr, ct_CUDA, 2 * NHalf * sizeof(Complex_d), hipMemcpyDeviceToHost);

    // outputFile.open("ct.txt", std::ios::out);
    // for(uint32_t i = 0; i < 2; i++)
    //     for(uint32_t j = 0; j < (N >> 1); j++)
    //         outputFile << "(" << ct_arr[i*(N >> 1) + j].real() << ", " << ct_arr[i*(N >> 1) + j].imag() << ")" << std::endl;
    // outputFile.close();

    // // Copy the dct_CUDA back to the host
    // Complex* dct_arr;
    // hipHostMalloc((void**)&dct_arr, digitsG2 * NHalf * sizeof(Complex));
    // hipMemcpy(dct_arr, dct_CUDA, digitsG2 * NHalf * sizeof(Complex_d), hipMemcpyDeviceToHost);

    // outputFile.open("dct.txt", std::ios::out);
    // for(uint32_t i = 0; i < digitsG2; i++)
    //     for(uint32_t j = 0; j < (N >> 1); j++)
    //         outputFile << "(" << dct_arr[i*(N >> 1) + j].real() << ", " << dct_arr[i*(N >> 1) + j].imag() << ")" << std::endl;
    // outputFile.close();
}

void AddToAccCGGI_CUDA(const std::shared_ptr<RingGSWCryptoParams> params, const std::vector<NativeVector>& a, 
        std::vector<std::vector<std::vector<Complex>>>& acc_d, std::string mode)
{   
    /* Parameters Set */
    uint32_t N            = params->GetN();
    uint32_t NHalf     = N >> 1;
    uint32_t digitsG2 = params->GetDigitsG() << 1;
    uint32_t arch = gpuInfoList[0].major * 100 + gpuInfoList[0].minor * 10;

    /* Determine template of AddToAccCGGI_CUDA_core */
    switch (arch){
        case 700: // V100
            switch (NHalf){
                case 512:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<700, 512, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<700, 512, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<700, 512, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<700, 512, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<700, 512, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<700, 512, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 1024:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<700, 1024, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<700, 1024, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<700, 1024, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<700, 1024, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<700, 1024, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<700, 1024, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 2048:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<700, 2048, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<700, 2048, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<700, 2048, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<700, 2048, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<700, 2048, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<700, 2048, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                default:
                    std::cerr << "Unsupported N\n";
                    exit(1);
            }
            break;
        case 800: // A100
            switch (NHalf){
                case 512:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<800, 512, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<800, 512, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<800, 512, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<800, 512, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<800, 512, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<800, 512, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 1024:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<800, 1024, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<800, 1024, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<800, 1024, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<800, 1024, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<800, 1024, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<800, 1024, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 2048:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<800, 2048, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<800, 2048, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<800, 2048, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<800, 2048, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<800, 2048, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<800, 2048, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                default:
                    std::cerr << "Unsupported N\n";
                    exit(1);
            }
            break;
        case 860: // RTX30 series
            switch (NHalf){
                case 512:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<860, 512, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<860, 512, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<860, 512, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<860, 512, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<860, 512, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<860, 512, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 1024:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<860, 1024, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<860, 1024, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<860, 1024, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<860, 1024, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<860, 1024, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<860, 1024, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 2048:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<860, 2048, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<860, 2048, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<860, 2048, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<860, 2048, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<860, 2048, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<860, 2048, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                default:
                    std::cerr << "Unsupported N\n";
                    exit(1);
            }
            break;
        case 890: // RTX40 series
            switch (NHalf){
                case 512:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<890, 512, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<890, 512, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<890, 512, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<890, 512, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<890, 512, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<890, 512, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 1024:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<890, 1024, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<890, 1024, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<890, 1024, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<890, 1024, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<890, 1024, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<890, 1024, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 2048:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<890, 2048, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<890, 2048, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<890, 2048, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<890, 2048, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<890, 2048, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<890, 2048, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                default:
                    std::cerr << "Unsupported N\n";
                    exit(1);
            }
            break;
        case 900: // H100
            switch (NHalf){
                case 512:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<900, 512, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<900, 512, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<900, 512, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<900, 512, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<900, 512, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<900, 512, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 1024:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<900, 1024, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<900, 1024, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<900, 1024, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<900, 1024, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<900, 1024, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<900, 1024, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                case 2048:
                    switch (digitsG2){
                        case 2:
                            AddToAccCGGI_CUDA_core<900, 2048, 2>(params, a, acc_d, mode);
                            break;
                        case 4:
                            AddToAccCGGI_CUDA_core<900, 2048, 4>(params, a, acc_d, mode);
                            break;
                        case 6:
                            AddToAccCGGI_CUDA_core<900, 2048, 6>(params, a, acc_d, mode);
                            break;
                        case 8:
                            AddToAccCGGI_CUDA_core<900, 2048, 8>(params, a, acc_d, mode);
                            break;
                        case 10:
                            AddToAccCGGI_CUDA_core<900, 2048, 10>(params, a, acc_d, mode);
                            break;
                        case 12:
                            AddToAccCGGI_CUDA_core<900, 2048, 12>(params, a, acc_d, mode);
                            break;
                        default:
                            std::cerr << "Unsupported digitsG\n";
                            exit(1);
                    }
                    break;
                default:
                    std::cerr << "Unsupported N\n";
                    exit(1);
            }
            break;
        default:
            std::cerr << "Unsupported GPU architecture\n";
            exit(1);
    }
}

template<uint32_t arch, uint32_t FFT_dimension, uint32_t FFT_num>
void AddToAccCGGI_CUDA_core(const std::shared_ptr<RingGSWCryptoParams> params, const std::vector<NativeVector>& a, 
        std::vector<std::vector<std::vector<Complex>>>& acc_d, std::string mode)
{   
    /* parameters set */
    auto mod        = a[0].GetModulus();
    uint32_t modInt = mod.ConvertToInt();
    auto Q            = params->GetQ();
    NativeInteger QHalf = Q >> 1;
    NativeInteger::SignedNativeInt Q_int = Q.ConvertToInt();
    uint32_t N         = params->GetN();
    uint32_t NHalf     = N >> 1;
    uint32_t n =  a[0].GetLength();
    uint32_t M      = 2 * params->GetN();
    uint32_t digitsG2 = params->GetDigitsG() << 1;

    int bootstrap_num = acc_d.size();
    int SM_count = gpuInfoList[0].multiprocessorCount;

    /* Configure cuFFTDx */
    using FFT     = decltype(cufftdx::Block() + cufftdx::Size<FFT_dimension>() + cufftdx::Type<cufftdx::fft_type::c2c>() + cufftdx::Direction<cufftdx::fft_direction::forward>() + cufftdx::ElementsPerThread<8>() +
                        cufftdx::Precision<double>() + cufftdx::FFTsPerBlock<FFT_num>() + cufftdx::SM<arch>());

    using IFFT     = decltype(cufftdx::Block() + cufftdx::Size<FFT_dimension>() + cufftdx::Type<cufftdx::fft_type::c2c>() + cufftdx::Direction<cufftdx::fft_direction::inverse>() + cufftdx::ElementsPerThread<8>() +
                            cufftdx::Precision<double>() + cufftdx::FFTsPerBlock<2>() + cufftdx::SM<arch>());

    using FFT_multi      = decltype(cufftdx::Block() + cufftdx::Size<FFT_dimension>() + cufftdx::Type<cufftdx::fft_type::c2c>() + cufftdx::Direction<cufftdx::fft_direction::forward>() +
                            cufftdx::Precision<double>() + cufftdx::FFTsPerBlock<2>() + cufftdx::SM<arch>());

    using IFFT_multi     = decltype(cufftdx::Block() + cufftdx::Size<FFT_dimension>() + cufftdx::Type<cufftdx::fft_type::c2c>() + cufftdx::Direction<cufftdx::fft_direction::inverse>() +
                            cufftdx::Precision<double>() + cufftdx::FFTsPerBlock<2>() + cufftdx::SM<arch>());

    /* Check whether block size exceeds cuda limitation */
    if(mode == "SINGLE"){
        if((NHalf / FFT::elements_per_thread * digitsG2) > gpuInfoList[0].maxThreadsPerBlock){
            std::cerr << "Exceed Maximum blocks per threads (" << gpuInfoList[0].maxThreadsPerBlock << ")\n";
            std::cerr << "Using " << (NHalf / FFT::elements_per_thread * digitsG2) << " threads" << ")\n";
            std::cerr << "NHalf: " << NHalf << "FFT::elements_per_thread: " << FFT::elements_per_thread << "digitsG2: " << digitsG2 << ")\n";
            exit(1);
        }
    }
    else if(mode == "MULTI"){
        if((NHalf / FFT_multi::elements_per_thread * 2) > gpuInfoList[0].maxThreadsPerBlock){
            std::cerr << "Exceed Maximum blocks per threads (" << gpuInfoList[0].maxThreadsPerBlock << ")\n";
            std::cerr << "Using " << (NHalf / FFT_multi::elements_per_thread * digitsG2) << " threads" << ")\n";
            std::cerr << "NHalf: " << NHalf << "FFT::elements_per_thread: " << FFT_multi::elements_per_thread << ")\n";
            exit(1);
        }
    }

    /* Check whether shared memory size exceeds cuda limitation */
    if(mode == "SINGLE"){
        if(FFT::shared_memory_size > gpuInfoList[0].sharedMemoryPerBlock){
            std::cerr << "Exceed Maximum sharedMemoryPerBlock ("<< gpuInfoList[0].sharedMemoryPerBlock << ")\n";
            std::cerr << "Declare "<< FFT::shared_memory_size << " now" << "\n";
            exit(1);
        }
    }
    else if(mode == "MULTI"){
        if(FFT_multi::shared_memory_size > gpuInfoList[0].sharedMemoryPerBlock){
            std::cerr << "Exceed Maximum sharedMemoryPerBlock ("<< gpuInfoList[0].sharedMemoryPerBlock << ")\n";
            std::cerr << "Declare "<< FFT_multi::shared_memory_size << " now" << "\n";
            exit(1);
        }
    }

    /* Initialize a_arr */
    uint64_t* a_arr;
    hipHostMalloc((void**)&a_arr, bootstrap_num * n * sizeof(uint64_t));
    for (int s = 0; s < bootstrap_num; s++)
        for (size_t i = 0; i < n; ++i)
            a_arr[s*n + i] = (mod.ModSub(a[s][i], mod) * (M / modInt)).ConvertToInt();

    /* Initialize acc_d_arr */
    Complex* acc_d_arr;
    hipHostMalloc((void**)&acc_d_arr, bootstrap_num * 2 * NHalf * sizeof(Complex));
    for (int s = 0; s < bootstrap_num; s++)
        for(int i = 0; i < 2; i++)
            for(int j = 0; j < NHalf; j++)
                acc_d_arr[s*2*NHalf + i*NHalf + j] = Complex(acc_d[s][i][j].real(), acc_d[s][i][j + NHalf].real());

    /* Measure GPU bootstrapping time */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    if(mode == "SINGLE"){
        for (int s = 0; s < bootstrap_num; s++) {
            hipMemcpyAsync(a_CUDA + (s % SM_count)*n, a_arr + s*n, n * sizeof(uint64_t), hipMemcpyHostToDevice, streams[s % SM_count]);
            hipMemcpyAsync(acc_CUDA + (s % SM_count)*2*NHalf, acc_d_arr + s*2*NHalf, 2 * NHalf * sizeof(Complex_d), hipMemcpyHostToDevice, streams[s % SM_count]);
            bootstrappingSingleBlock<FFT, IFFT><<<1, FFT::block_dim, FFT::shared_memory_size, streams[s % SM_count]>>>
                (acc_CUDA + (s % SM_count)*2*NHalf, ct_CUDA + (s % SM_count)*2*NHalf, dct_CUDA + (s % SM_count)*digitsG2*NHalf, a_CUDA + (s % SM_count)*n, 
                monomial_CUDA, twiddleTable_CUDA, params_CUDA, GINX_bootstrappingKey_CUDA);
            hipMemcpyAsync(acc_d_arr + s*2*NHalf, acc_CUDA + (s % SM_count)*2*NHalf, 2 * NHalf * sizeof(Complex_d), hipMemcpyDeviceToHost, streams[s % SM_count]);
        }
    }
    else if(mode == "MULTI"){
        Complex_d* acc_CUDA_offset, *ct_CUDA_offset, *dct_CUDA_offset;
        uint64_t* a_CUDA_offset;
        for (int s = 0; s < bootstrap_num; s++) {
            acc_CUDA_offset = acc_CUDA + (s % SM_count)*2*NHalf;
            ct_CUDA_offset = ct_CUDA + (s % SM_count)*2*NHalf;
            dct_CUDA_offset = dct_CUDA + (s % SM_count)*digitsG2*NHalf;
            a_CUDA_offset = a_CUDA + (s % SM_count)*n;
            hipMemcpyAsync(a_CUDA + (s % SM_count)*n, a_arr + s*n, n * sizeof(uint64_t), hipMemcpyHostToDevice, streams[s % SM_count]);
            hipMemcpyAsync(acc_CUDA + (s % SM_count)*2*NHalf, acc_d_arr + s*2*NHalf, 2 * NHalf * sizeof(Complex_d), hipMemcpyHostToDevice, streams[s % SM_count]);
            void *kernelArgs[] = {(void *)&acc_CUDA_offset, (void *)&ct_CUDA_offset, (void *)&dct_CUDA_offset, (void *)&a_CUDA_offset, 
                (void *)&monomial_CUDA, (void *)&twiddleTable_CUDA, (void *)&params_CUDA, (void *)&GINX_bootstrappingKey_CUDA};
            hipLaunchCooperativeKernel((void*)(bootstrappingMultiBlock<FFT_multi, IFFT_multi>), digitsG2/2, FFT_multi::block_dim, 
                kernelArgs, FFT_multi::shared_memory_size, streams[s % SM_count]);
            hipMemcpyAsync(acc_d_arr + s*2*NHalf, acc_CUDA + (s % SM_count)*2*NHalf, 2 * NHalf * sizeof(Complex_d), hipMemcpyDeviceToHost, streams[s % SM_count]);
        }
    }
    // CUDA_CHECK_AND_EXIT(hipPeekAtLastError());
    // CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << bootstrap_num << " Bootstrapping GPU time : " << milliseconds << " ms\n";
    hipEventDestroy(start);
    hipEventDestroy(stop);

    /* Copy acc_d_arr back to acc_d */
    for (int s = 0; s < bootstrap_num; s++) {
        for(int i = 0; i < 2; i++){
            for(int j = 0; j < NHalf; j++){
                acc_d[s][i][j] = Complex(acc_d_arr[s*2*NHalf + i*NHalf + j].real(), 0);
                acc_d[s][i][j + NHalf] = Complex(acc_d_arr[s*2*NHalf + i*NHalf + j].imag(), 0);
            }
        }
    }

    /* Free memory */     
    hipHostFree(a_arr);
    hipHostFree(acc_d_arr);
}

void MKMSwitch_CUDA(const std::shared_ptr<LWECryptoParams> params, std::shared_ptr<std::vector<LWECiphertext>> ctExt, NativeInteger Q1, NativeInteger Q2)
{
    /* parameters set */
    uint32_t n        = params->Getn();
    uint32_t N        = params->GetN();
    NativeInteger q   = params->Getq().ConvertToInt();
    int64_t q_int = q.ConvertToInt();
    NativeInteger Q   = params->GetQ().ConvertToInt();
    int64_t Q_int = Q.ConvertToInt();
    uint32_t baseKS   = params->GetBaseKS();
    uint32_t digitCountKS = (uint32_t)std::ceil(log(Q1.ConvertToDouble()) / log(static_cast<double>(baseKS)));
    
    int bootstrap_num = ctExt->size();
    int SM_count = gpuInfoList[0].multiprocessorCount;

    /* Initialize paramsMKM_CUDA */
    uint64_t *paramters;
    hipHostMalloc((void**)&paramters, 8 * sizeof(uint64_t));
    paramters[0] = n;
    paramters[1] = N;
    paramters[2] = static_cast<uint64_t>(q_int);
    paramters[3] = static_cast<uint64_t>(Q_int);
    paramters[4] = baseKS;
    paramters[5] = digitCountKS;
    paramters[6] = static_cast<uint64_t>(Q1.ConvertToInt());
    paramters[7] = static_cast<uint64_t>(Q2.ConvertToInt());
    // Bring paramsMKM_CUDA to GPU
    uint64_t *paramsMKM_CUDA;
    hipMalloc(&paramsMKM_CUDA, 8 * sizeof(uint64_t));
    hipMemcpy(paramsMKM_CUDA, paramters, 8 * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipHostFree(paramters);

    /* Initialize ctExt_CUDA */
    uint64_t* ctExt_CUDA;
    hipMalloc((void**)&ctExt_CUDA, bootstrap_num * (N + 1) * sizeof(uint64_t));
    uint64_t* ctExt_host;
    hipHostMalloc((void**)&ctExt_host, bootstrap_num * (N + 1) * sizeof(uint64_t));
    for (int s = 0; s < bootstrap_num; s++){
        // A
        for(int i = 0; i < N; i++)
            ctExt_host[s*(N + 1) + i] = static_cast<uint64_t>((*ctExt)[s]->GetA()[i].ConvertToInt());
        // B
        ctExt_host[s*(N + 1) + N] = static_cast<uint64_t>((*ctExt)[s]->GetB().ConvertToInt());
    }

    /* Measure GPU time */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int s = 0; s < bootstrap_num; s++) {
        hipMemcpyAsync(ctExt_CUDA + s*(N + 1), ctExt_host + s*(N + 1), (N + 1) * sizeof(uint64_t), hipMemcpyHostToDevice, streams[s % SM_count]);
        MKMSwitchKernel<<<1, 512, (n + 1) * sizeof(uint64_t), streams[s % SM_count]>>>(ctExt_CUDA + s*(N + 1), keySwitchingkey_CUDA, paramsMKM_CUDA);
        hipMemcpyAsync(ctExt_host + s*(N + 1), ctExt_CUDA + s*(N + 1), (N + 1) * sizeof(uint64_t), hipMemcpyDeviceToHost, streams[s % SM_count]);
    }
    // CUDA_CHECK_AND_EXIT(hipPeekAtLastError());
    // CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << bootstrap_num << " MKMSwitching GPU time : " << milliseconds << " ms\n";
    hipEventDestroy(start);
    hipEventDestroy(stop);

    /* Copy ctExt_host back to ctExt */
    for (int s = 0; s < bootstrap_num; s++){
        // A
        NativeVector a(n, Q2);
        for(int i = 0; i < n; i++)
            a[i] = ctExt_host[s*(N + 1) + i];
        // B
        NativeInteger b (ctExt_host[s*(N + 1) + n]);

        (*ctExt)[s] = std::make_shared<LWECiphertextImpl>(LWECiphertextImpl(std::move(a), b));
    }

    /* Free memory */     
    hipHostFree(ctExt_host);
}

};  // namespace lbcrypto


//     /* Measure GPU bootstrapping time */
//     hipEvent_t start, stop;
//     hipEventCreate(&start);
//     hipEventCreate(&stop);
//     hipEventRecord(start);

//     for (int s = 0; s < bootstrap_num; s++) {
//         // Bring input ciphertext to GPU
//         hipMalloc(&input_dev[s], 2 * fft_size * sizeof(Complex_d));
//         hipMemcpyAsync(input_dev[s], input[s], 2 * fft_size * sizeof(Complex_d), hipMemcpyHostToDevice, streams[s]);

//         bootstrapping_Baseline<FFT, IFFT><<<1, FFT::block_dim, 81920, streams[s]>>>(input_dev[s], bootstrappingKey_dev, twiddleTable);

//         // Copy the result back to the host
//         hipMemcpyAsync(input[s], input_dev[s], 2 * fft_size * sizeof(Complex_d), hipMemcpyDeviceToHost, streams[s]);
//     }

//     hipEventRecord(stop);
//     hipEventSynchronize(stop);
//     float milliseconds = 0;
//     hipEventElapsedTime(&milliseconds, start, stop);
//     std::cout << bootstrap_num << " Bootstrapping GPU time : " << milliseconds << " ms\n";
//     hipEventDestroy(start);
//     hipEventDestroy(stop);

//     /* Free memory */     
//     for (int s = 0; s < bootstrap_num; s++) {
//         hipFree(input_dev[s]);
//         hipHostFree(input[s]);
//         hipStreamDestroy(streams[s]);
//     }
//     hipHostFree(bootstrappingKey);
// }