#include "hip/hip_runtime.h"
#include "lwe-operation.cuh"

// cuBLAS error checking macro
#define CUBLAS_ERROR_CHECK(call) \
do { \
    hipblasStatus_t err = call; \
    if (err != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "cuBLAS error: " << err << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    } \
} while(0)

namespace lbcrypto {
    // Definition of the static member variables
    cublasXtHandle_t GPULWEOperation::handle;

    std::shared_ptr<std::vector<LWECiphertext>> GPULWEOperation::CiphertextMulMatrix_CUDA(const std::shared_ptr<BinFHECryptoParams> params, 
            const std::vector<LWECiphertext>& ct, const std::vector<std::vector<int64_t>>& matrix){
        
        /* Error check */
        if (ct.empty()) {
            std::cerr << "Input ciphertexts are empty." << std::endl;
            exit(EXIT_FAILURE);
        }
        if (matrix.empty()) {
            std::cerr << "Input matrix is empty." << std::endl;
            exit(EXIT_FAILURE);
        }
        if (ct.size() != matrix.size()) {
            std::cerr << "The number of rows of the matrix must be equal to the number of input ciphertexts." << std::endl;
            exit(EXIT_FAILURE);
        }

        /* Parameters Set */
        uint32_t M                  = params->GetLWEParams()->Getn() + 1;
        uint32_t N                  = matrix[0].size();
        uint32_t K                  = ct.size();
        uint64_t qKS                = params->GetLWEParams()->GetqKS().ConvertToInt();
        uint32_t n                  = params->GetLWEParams()->Getn();
        
        /* Allocate matrices on host */
        double *h_A = (double *)malloc(M * K * sizeof(double));
        double *h_B = (double *)malloc(K * N * sizeof(double));
        double *h_C = (double *)malloc(M * N * sizeof(double));
        
        /* Initialize matrices on host */
        for (int i = 0; i < K; ++i){
            NativeVector& ct_A = ct[i]->GetA();
            for (int j = 0; j < (M - 1); ++j){
                h_A[M*i + j] = ct_A[j].ConvertToDouble();
            }
            h_A[M*i + (M - 1)] = ct[i]->GetB().ConvertToDouble();
        }
        for (int i = 0; i < K; ++i)
            for (int j = 0; j < N; ++j)
                h_B[N*i + j] = static_cast<double>(matrix[i][j]);

        /* Perform matrix multiplication on device */
        const double alpha = 1.0f, beta = 0.0f;
        hipblasOperation_t transa = HIPBLAS_OP_N, transb = HIPBLAS_OP_T;
        CUBLAS_ERROR_CHECK(cublasXtDgemm(handle, transa, transb, M, N, K, &alpha, h_A, M, h_B, N, &beta, h_C, M));

        /* Serialize result matrix to ciphertexts */
        auto ct_res = std::make_shared<std::vector<LWECiphertext>> (N);
        for (int i = 0; i < N; ++i) {
            // A
            NativeVector a(n, qKS);
            for(int j = 0; j < (M-1); j++)
                a[j] = static_cast<uint64_t>(fmod(h_C[M*i + j], qKS));
            // B
            NativeInteger b (static_cast<uint64_t>(fmod(h_C[M*i + (M-1)], qKS)));

            (*ct_res)[i] = std::make_shared<LWECiphertextImpl>(LWECiphertextImpl(std::move(a), b));
        }

        /* Free host memory */
        free(h_A);
        free(h_B);
        free(h_C);
        
        return ct_res;
    }

    void GPULWEOperation::GPUSetup(int numGPUs){
        /* Setting up available GPU INFO */
        int deviceCount;
        hipGetDeviceCount(&deviceCount);

        if (deviceCount == 0) {
            std::cerr << "No CUDA devices found." << std::endl;
            return;
        }

        /* Determine the number of GPUs to use*/
        int GPUcount;
        if(numGPUs > 0 && numGPUs <= deviceCount) GPUcount = numGPUs;
        else GPUcount = deviceCount;

        /* Set the device IDs to use */
        int* devices = (int *)malloc(GPUcount * sizeof(int));
        for(int i = 0; i < GPUcount; i++){
            devices[i] = i;
        }

        /* Initialize cuBLAS Xt */
        CUBLAS_ERROR_CHECK(cublasXtCreate(&handle));
        CUBLAS_ERROR_CHECK(cublasXtDeviceSelect(handle, GPUcount, devices));

        /* Free devices array */
        free(devices);
    }

    void GPULWEOperation::GPUClean(){
        CUBLAS_ERROR_CHECK(cublasXtDestroy(handle));
    }

}; // namespace lbcrypto
