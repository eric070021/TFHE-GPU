#include "hip/hip_runtime.h"
#include "lwe-operation.cuh"

// cuBLAS error checking macro
#define CUBLAS_ERROR_CHECK(call) \
do { \
    hipblasStatus_t err = call; \
    if (err != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "cuBLAS error: " << err << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    } \
} while(0)

namespace lbcrypto {
    // Definition of the static member variables
    hipblasHandle_t GPULWEOperation::handle;

    __global__ void applyFmod(double* matrix, int size, double divisor) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;

        if (index < size) {
            matrix[index] = fmod(matrix[index], divisor);
        }
    }

    std::shared_ptr<std::vector<LWECiphertext>> GPULWEOperation::CiphertextMulMatrix_CUDA(const std::shared_ptr<BinFHECryptoParams> params, 
            const std::vector<LWECiphertext>& ct, const std::vector<std::vector<int64_t>>& matrix){
        
        /* Error check */
        if (ct.empty()) {
            std::cerr << "Input ciphertexts are empty." << std::endl;
            exit(EXIT_FAILURE);
        }
        if (matrix.empty()) {
            std::cerr << "Input matrix is empty." << std::endl;
            exit(EXIT_FAILURE);
        }
        if (ct.size() != matrix.size()) {
            std::cerr << "The number of rows of the matrix must be equal to the number of input ciphertexts." << std::endl;
            exit(EXIT_FAILURE);
        }

        /* Parameters Set */
        uint32_t M                  = params->GetLWEParams()->Getn() + 1;
        uint32_t N                  = matrix[0].size();
        uint32_t K                  = ct.size();
        uint64_t qKS                = params->GetLWEParams()->GetqKS().ConvertToInt();
        uint32_t n                  = params->GetLWEParams()->Getn();
        
        /* Allocate matrices on host */
        double *h_A = (double *)malloc(M * K * sizeof(double));
        double *h_B = (double *)malloc(K * N * sizeof(double));
        double *h_C = (double *)malloc(M * N * sizeof(double));
        
        /* Initialize matrices on host */
        for (int i = 0; i < K; ++i){
            NativeVector& ct_A = ct[i]->GetA();
            for (int j = 0; j < (M - 1); ++j){
                h_A[M*i + j] = ct_A[j].ConvertToDouble();
            }
            h_A[M*i + (M - 1)] = ct[i]->GetB().ConvertToDouble();
        }
        for (int i = 0; i < K; ++i)
            for (int j = 0; j < N; ++j)
                h_B[N*i + j] = static_cast<double>(matrix[i][j]);

        /* Set the device to use */
        hipSetDevice(0);
        
        /* Allocate memory on the GPU */
        double *d_A, *d_B, *d_C;
        hipMalloc(&d_A, M * K * sizeof(double));
        hipMalloc(&d_B, K * N * sizeof(double));
        hipMalloc(&d_C, M * N * sizeof(double));

        /* Copy input matrices from host to GPU */
        hipMemcpy(d_A, h_A, M * K * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, K * N * sizeof(double), hipMemcpyHostToDevice);

        /* Perform matrix multiplication on device */
        const double alpha = 1.0f, beta = 0.0f;
        hipblasOperation_t transa = HIPBLAS_OP_N, transb = HIPBLAS_OP_T;
        CUBLAS_ERROR_CHECK(hipblasDgemm(handle, transa, transb, M, N, K, &alpha, d_A, M, d_B, N, &beta, d_C, M));

        /* Launch the kernel */
        int blockSize = 256; // Choose an appropriate block size
        int gridSize = (M * N + blockSize - 1) / blockSize;
        applyFmod<<<gridSize, blockSize>>>(d_C, M * N, static_cast<double>(qKS));
        hipDeviceSynchronize();

        /* Copy result matrix from GPU to host */
        hipMemcpy(h_C, d_C, M * N * sizeof(double), hipMemcpyDeviceToHost);

        /* Serialize result matrix to ciphertexts */
        auto ct_res = std::make_shared<std::vector<LWECiphertext>> (N);
        for (int i = 0; i < N; ++i) {
            // A
            NativeVector a(n, qKS);
            for(int j = 0; j < (M-1); j++)
                a[j] = static_cast<uint64_t>(h_C[M*i + j]);
            // B
            NativeInteger b (static_cast<uint64_t>(h_C[M*i + (M-1)]));

            (*ct_res)[i] = std::make_shared<LWECiphertextImpl>(LWECiphertextImpl(std::move(a), b));
        }

        /* Free host memory */
        free(h_A);
        free(h_B);
        free(h_C);

        /* Free memory on GPU */
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        
        return ct_res;
    }

    void GPULWEOperation::GPUSetup(int numGPUs){
        hipSetDevice(0);
        hipblasCreate(&handle);
    }

    void GPULWEOperation::GPUClean(){
        hipSetDevice(0);
        hipblasDestroy(handle);
    }

}; // namespace lbcrypto
